#include "cuda_helper.hpp"

#include "my_utils.hpp"

int configureGPU(const int dev, hipblasHandle_t &handle,
#ifdef USE_CUSOLVER
                 hipsolverHandle_t &csh,
#endif /* USE_CUSOLVER */
                 const hipStream_t s) throw()
{
  assert(dev >= 0);
  hipDeviceProp_t cdp;
  CUDA_CALL(hipGetDeviceProperties(&cdp, dev));
#ifndef USE_CUSOLVER
  if (WARP_SZ != static_cast<unsigned>(cdp.warpSize)) {
    (void)snprintf(err_msg, err_msg_size, "CUDA Device %d has %d threads in a warp, must be %u", dev, cdp.warpSize, WARP_SZ);
    DIE(err_msg);
  }
#endif /* !USE_CUSOLVER */
  CUDA_CALL(hipSetDevice(dev));
#ifndef USE_CUSOLVER
  CUDA_CALL(hipDeviceSetCacheConfig(hipFuncCachePreferShared));
#endif /* !USE_CUSOLVER */
  CUDA_CALL(hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte));

  CUBLAS_CALL(hipblasCreate(&handle));
  CUBLAS_CALL(hipblasSetStream(handle, s));
  CUBLAS_CALL(hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST));
#ifdef USE_CUSOLVER
  CUSOLVER_CALL(hipsolverDnCreate(&csh));
  CUSOLVER_CALL(hipsolverSetStream(csh, s));
#else /* !USE_CUSOLVER */
  CUBLAS_CALL(hipblasSetAtomicsMode(handle, HIPBLAS_ATOMICS_NOT_ALLOWED));
#endif /* ?USE_CUSOLVER */
  return (cdp.major * 10 + cdp.minor);
}

void freeGPU(hipblasHandle_t &handle
#ifdef USE_CUSOLVER
             , hipsolverHandle_t &csh
#endif /* USE_CUSOLVER */
             ) throw()
{
#ifdef USE_CUSOLVER
  CUSOLVER_CALL(hipsolverDnDestroy(csh));
#endif /* USE_CUSOLVER */
  CUBLAS_CALL(hipblasDestroy(handle));
#if (defined(PROFILE) && (PROFILE != 0))
  CUDA_CALL(hipDeviceReset());
#endif /* ?PROFILE */
}

void cuda_prof_start() throw()
{
#if (defined(PROFILE) && (PROFILE != 0))
  CUDA_CALL(hipProfilerStart());
#endif /* ?PROFILE */
}

void cuda_prof_stop() throw()
{
#if (defined(PROFILE) && (PROFILE != 0))
  CUDA_CALL(hipProfilerStop());
#endif /* ?PROFILE */
}
