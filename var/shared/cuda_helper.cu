#include "cuda_helper.hpp"

#include "my_utils.hpp"

int configureGPU(const int dev, hipblasHandle_t &handle) throw()
{
  assert(dev >= 0);
  hipDeviceProp_t cdp;
  CUDA_CALL(hipGetDeviceProperties(&cdp, dev));
  if (WARP_SZ != static_cast<unsigned>(cdp.warpSize)) {
    (void)snprintf(err_msg, err_msg_size, "CUDA Device %d has %d threads in a warp, must be %u", dev, cdp.warpSize, WARP_SZ);
    DIE(err_msg);
  }

  CUDA_CALL(hipSetDevice(dev));
  CUDA_CALL(hipDeviceSetCacheConfig(hipFuncCachePreferShared));
  CUDA_CALL(hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte));

  CUBLAS_CALL(hipblasCreate(&handle));
  CUBLAS_CALL(hipblasSetStream(handle, static_cast<hipStream_t>(NULL)));
  CUBLAS_CALL(hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST));
  CUBLAS_CALL(hipblasSetAtomicsMode(handle, HIPBLAS_ATOMICS_NOT_ALLOWED));

  return (cdp.major * 10 + cdp.minor);
}

void freeGPU(hipblasHandle_t &handle) throw()
{
  CUBLAS_CALL(hipblasDestroy(handle));
#if (defined(PROFILE) && (PROFILE != 0))
  CUDA_CALL(hipDeviceReset());
#else /* !PROFILE || PROFILE == 0 */
  CUDA_CALL(hipDeviceSynchronize());
#endif /* ?PROFILE */
}

void cuda_prof_start() throw()
{
#if (defined(PROFILE) && (PROFILE != 0))
  CUDA_CALL(hipProfilerStart());
#endif /* ?PROFILE */
}

void cuda_prof_stop() throw()
{
#if (defined(PROFILE) && (PROFILE != 0))
  CUDA_CALL(hipProfilerStop());
#endif /* ?PROFILE */
}
