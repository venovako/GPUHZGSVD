#include "cuda_memory_helper.hpp"

int main(int argc, char *argv[])
{
  if ((argc < 5) || (argc > 6)) {
    (void)fprintf(stderr, "%s mF mG n fn [dev]\n", *argv);
    (void)fflush(stderr);
    return EXIT_FAILURE;
  }
  const unsigned mF = static_cast<unsigned>(atoi(argv[1]));
  if (!mF)
    return EXIT_FAILURE;
  const unsigned mG = static_cast<unsigned>(atoi(argv[2]));
  if (!mG)
    return EXIT_FAILURE;
  const unsigned n = static_cast<unsigned>(atoi(argv[3]));
  if (!n)
    return EXIT_FAILURE;
  if (mF < n)
    return EXIT_FAILURE;
  if (mG < n)
    return EXIT_FAILURE;
  const int dev = ((argc == 6) ? atoi(argv[5]) : 0);
  hipblasHandle_t cbh = 0;
  hipsolverHandle_t csh = 0;
  const int dcc = configureGPU(dev, cbh, csh);
#ifndef NDEBUG
  (void)fprintf(stderr, "GPU device %d is of compute capability %d.\n", dev, dcc);
  (void)fflush(stderr);
#endif /* !NDEBUG */
  char *const fn = static_cast<char*>(calloc(strlen(argv[4]) + 3u, sizeof(char)));
  SYSP_CALL(fn);
  FILE *const f = fopen(strcat(strcpy(fn, argv[4]), ".Y"), "rb");
  SYSP_CALL(f);
  size_t ldhF = mF;
  double *const hF = allocHostMtx<double>(ldhF, mF, n, true);
  SYSP_CALL(hF);
  SYSI_CALL(fread_bycol(f, mF, n, hF, ldhF));
  SYSI_CALL(fclose(f));
  FILE *const g = fopen(strcat(strcpy(fn, argv[4]), ".W"), "rb");
  SYSP_CALL(g);
  size_t ldhG = mG;
  double *const hG = allocHostMtx<double>(ldhG, mG, n, true);
  SYSP_CALL(hG);
  SYSI_CALL(fread_bycol(g, mG, n, hG, ldhG));
  SYSI_CALL(fclose(g));
  size_t lddF = mF;
  double *const dF = allocDeviceMtx<double>(lddF, mF, n, true);
  SYSP_CALL(dF);
  size_t lddG = mG;
  double *const dG = allocDeviceMtx<double>(lddG, mG, n, true);
  SYSP_CALL(dG);
  size_t lddA = n;
  double *const dA = allocDeviceMtx<double>(lddA, n, n, true);
  SYSP_CALL(dA);
  size_t lddB = n;
  double *const dB = allocDeviceMtx<double>(lddB, n, n, true);
  SYSP_CALL(dB);
  double *const dW = allocDeviceVec<double>(n + 1u);
  SYSP_CALL(dW);
  int *const dinfo = reinterpret_cast<int*>(dW + n);
  int lwork = 0;
  CUSOLVER_CALL(hipsolverDnDsygvd_bufferSize(csh, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_LOWER, n, dA, lddA, dB, lddB, dW, &lwork));
  if (lwork < 0)
    return EXIT_FAILURE;
  (void)fprintf(stdout, "%d,", lwork);
  (void)fflush(stdout);
  const size_t mM = ((mF >= mG) ? mF : mG);
  size_t lddM = ((lddF >= lddG) ? lddF : lddG);
  size_t n2 = static_cast<size_t>(n) * 2u;
  if ((lddM * n2) < lwork)
    n2 = ((static_cast<size_t>(lwork) + (lddM - 1u)) / lddM);
  double *const dwork = allocDeviceMtx<double>(lddM, mM, n2, true);
  SYSP_CALL(dwork);
  const size_t lddU = lddM;
  double *const dU = dwork;
  const size_t lddV = lddM;
  double *const dV = dwork + (lddM * n);
  CUDA_CALL(hipDeviceSynchronize());
  CUDA_CALL(hipMemcpy2D(dF, lddF * sizeof(double), hF, ldhF * sizeof(double), mF * sizeof(double), n, hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy2D(dG, lddG * sizeof(double), hG, ldhG * sizeof(double), mG * sizeof(double), n, hipMemcpyHostToDevice));
  const double alpha = 1.0;
  const double beta = 0.0;
  long long sw = 0ll, t = 0ll, tt = 0ll;
  CUDA_CALL(hipDeviceSynchronize());
  stopwatch_reset(sw);
  CUBLAS_CALL(hipblasDsyrk(cbh, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_T, n, mF, &alpha, dF, lddF, &beta, dA, lddA));
  CUBLAS_CALL(hipblasDsyrk(cbh, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_T, n, mG, &alpha, dG, lddG, &beta, dB, lddB));
  CUDA_CALL(hipDeviceSynchronize());
  (void)fprintf(stdout, "%lld,", (t = stopwatch_lap(sw)));
  (void)fflush(stdout);
  tt += t;
  CUSOLVER_CALL(hipsolverDnDsygvd(csh, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_LOWER, n, dA, lddA, dB, lddB, dW, dwork, lwork, dinfo));
  CUDA_CALL(hipDeviceSynchronize());
  (void)fprintf(stdout, "%lld,", (t = stopwatch_lap(sw)));
  (void)fflush(stdout);
  tt += t;
  CUBLAS_CALL(hipblasDgemm(cbh, HIPBLAS_OP_N, HIPBLAS_OP_N, mF, n, n, &alpha, dF, lddF, dA, lddA, &beta, dU, lddU));
  CUBLAS_CALL(hipblasDgemm(cbh, HIPBLAS_OP_N, HIPBLAS_OP_N, mG, n, n, &alpha, dG, lddG, dA, lddA, &beta, dV, lddV));
  CUDA_CALL(hipDeviceSynchronize());
  (void)fprintf(stdout, "%lld,", (t = stopwatch_lap(sw)));
  (void)fflush(stdout);
  tt += t;
  CUDA_CALL(hipFree(dG));
  CUDA_CALL(hipFree(dF));
  double *const hW = allocHostVec<double>(n + 1u);
  SYSP_CALL(hW);
  CUDA_CALL(hipMemcpy(hW, dW, (n + 1u) * sizeof(double), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy2D(hF, ldhF * sizeof(double), dU, lddU * sizeof(double), mF * sizeof(double), n, hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy2D(hG, ldhG * sizeof(double), dV, lddV * sizeof(double), mG * sizeof(double), n, hipMemcpyDeviceToHost));
  CUDA_CALL(hipDeviceSynchronize());
  CUDA_CALL(hipFree(dwork));
  CUDA_CALL(hipFree(dW));
  (void)fprintf(stdout, "%lld,%d\n", tt, *reinterpret_cast<const int*>(hW + n));
  (void)fflush(stdout);
  FILE *const l = fopen(strcat(strcpy(fn, argv[4]), ".L"), "wb");
  SYSP_CALL(l);
  SYSI_CALL(fwrite(hW, sizeof(double), n, l) != n);
  SYSI_CALL(fclose(l));
  CUDA_CALL(hipHostFree(hW));
  FILE *const v = fopen(strcat(strcpy(fn, argv[4]), ".V"), "wb");
  SYSP_CALL(v);
  SYSI_CALL(fresize(v, mG * (n * sizeof(double))));
  SYSI_CALL(fwrite_bycol(v, mG, n, hG, ldhG));
  SYSI_CALL(fclose(v));
  CUDA_CALL(hipMemcpy2D(hG, ldhG * sizeof(double), dB, lddB * sizeof(double), n * sizeof(double), n, hipMemcpyDeviceToHost));
  FILE *const u = fopen(strcat(strcpy(fn, argv[4]), ".U"), "wb");
  SYSP_CALL(u);
  SYSI_CALL(fresize(u, mF * (n * sizeof(double))));
  SYSI_CALL(fwrite_bycol(u, mF, n, hF, ldhF));
  SYSI_CALL(fclose(u));
  CUDA_CALL(hipMemcpy2D(hF, ldhF * sizeof(double), dA, lddA * sizeof(double), n * sizeof(double), n, hipMemcpyDeviceToHost));
  CUDA_CALL(hipDeviceSynchronize());
  CUDA_CALL(hipFree(dB));
  CUDA_CALL(hipFree(dA));
  for (unsigned j = 1u; j < n; ++j) {
    double *const Gj = hG + j * ldhG;
    for (unsigned i = 0u; i < j; ++i)
      Gj[i] = beta;
  }
  FILE *const b = fopen(strcat(strcpy(fn, argv[4]), ".B"), "wb");
  SYSP_CALL(b);
  SYSI_CALL(fresize(b, n * (n * sizeof(double))));
  SYSI_CALL(fwrite_bycol(b, n, n, hG, ldhG));
  SYSI_CALL(fclose(b));
  CUDA_CALL(hipHostFree(hG));
  FILE *const e = fopen(strcat(strcpy(fn, argv[4]), ".E"), "wb");
  SYSP_CALL(e);
  SYSI_CALL(fresize(e, n * (n * sizeof(double))));
  SYSI_CALL(fwrite_bycol(e, n, n, hF, ldhF));
  SYSI_CALL(fclose(e));
  CUDA_CALL(hipHostFree(hF));
  free(fn);
  freeGPU(cbh, csh);
  return EXIT_SUCCESS;
}
