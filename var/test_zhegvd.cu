#include "cuda_memory_helper.hpp"

int main(int argc, char *argv[])
{
  if ((argc < 5) || (argc > 6)) {
    (void)fprintf(stderr, "%s mF mG n fn [dev]\n", *argv);
    (void)fflush(stderr);
    return EXIT_FAILURE;
  }
  const unsigned mF = static_cast<unsigned>(atoi(argv[1]));
  if (!mF)
    return EXIT_FAILURE;
  const unsigned mG = static_cast<unsigned>(atoi(argv[2]));
  if (!mG)
    return EXIT_FAILURE;
  const unsigned n = static_cast<unsigned>(atoi(argv[3]));
  if (!n)
    return EXIT_FAILURE;
  if (mF < n)
    return EXIT_FAILURE;
  if (mG < n)
    return EXIT_FAILURE;
  const int dev = ((argc == 6) ? atoi(argv[5]) : 0);
  hipblasHandle_t cbh = 0;
  hipsolverHandle_t csh = 0;
  const int dcc = configureGPU(dev, cbh, csh);
#ifndef NDEBUG
  (void)fprintf(stderr, "GPU device %d is of compute capability %d.\n", dev, dcc);
  (void)fflush(stderr);
#endif /* !NDEBUG */
  char *const fn = static_cast<char*>(calloc(strlen(argv[4]) + 3u, sizeof(char)));
  SYSP_CALL(fn);
  FILE *const f = fopen(strcat(strcpy(fn, argv[4]), ".Y"), "rb");
  SYSP_CALL(f);
  size_t ldhF = mF;
  hipDoubleComplex *const hF = allocHostMtx<hipDoubleComplex>(ldhF, mF, n, true);
  SYSP_CALL(hF);
  SYSI_CALL(fread_bycol(f, mF, n, hF, ldhF));
  SYSI_CALL(fclose(f));
  FILE *const g = fopen(strcat(strcpy(fn, argv[4]), ".W"), "rb");
  SYSP_CALL(g);
  size_t ldhG = mG;
  hipDoubleComplex *const hG = allocHostMtx<hipDoubleComplex>(ldhG, mG, n, true);
  SYSP_CALL(hG);
  SYSI_CALL(fread_bycol(g, mG, n, hG, ldhG));
  SYSI_CALL(fclose(g));
  size_t lddA = n;
  hipDoubleComplex *const dA = allocDeviceMtx<hipDoubleComplex>(lddA, n, n, true);
  SYSP_CALL(dA);
  size_t lddB = n;
  hipDoubleComplex *const dB = allocDeviceMtx<hipDoubleComplex>(lddB, n, n, true);
  SYSP_CALL(dB);
  double *const dW = allocDeviceVec<double>(n + 1u);
  SYSP_CALL(dW);
  int *const dinfo = reinterpret_cast<int*>(dW + n);
  int lwork = 0;
  CUSOLVER_CALL(hipsolverDnZhegvd_bufferSize(csh, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_LOWER, n, dA, lddA, dB, lddB, dW, &lwork));
  if (lwork < 0)
    return EXIT_FAILURE;
  (void)fprintf(stdout, "%d,", lwork);
  (void)fflush(stdout);
  const unsigned mM = ((mF >= mG) ? mF : mG);
  size_t lddM = mM;
  size_t n2 = (static_cast<size_t>(n) * 2u);
  if ((lddM * n2) < lwork)
    n2 = ((static_cast<size_t>(lwork) + (lddM - 1u)) / lddM);
  hipDoubleComplex *const dwork = allocDeviceMtx<hipDoubleComplex>(lddM, mM, n2, true);
  SYSP_CALL(dwork);
  const size_t lddF = lddM;
  hipDoubleComplex *const dF = dwork;
  const size_t lddG = lddM;
  hipDoubleComplex *const dG = dwork + (lddM * n);
  CUDA_CALL(hipDeviceSynchronize());
  CUDA_CALL(hipMemcpy2D(dF, lddF * sizeof(hipDoubleComplex), hF, ldhF * sizeof(hipDoubleComplex), mF * sizeof(hipDoubleComplex), n, hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy2D(dG, lddG * sizeof(hipDoubleComplex), hG, ldhG * sizeof(hipDoubleComplex), mG * sizeof(hipDoubleComplex), n, hipMemcpyHostToDevice));
  const double alpha = 1.0;
  const double beta = 0.0;
  long long sw = 0ll;
  CUDA_CALL(hipDeviceSynchronize());
  stopwatch_reset(sw);
  CUBLAS_CALL(hipblasZherk(cbh, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_C, n, mF, &alpha, dF, lddF, &beta, dA, lddA));
  CUBLAS_CALL(hipblasZherk(cbh, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_C, n, mG, &alpha, dG, lddG, &beta, dB, lddB));
  CUDA_CALL(hipDeviceSynchronize());
  (void)fprintf(stdout, "%lld,", stopwatch_lap(sw));
  (void)fflush(stdout);
  CUSOLVER_CALL(hipsolverDnZhegvd(csh, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_LOWER, n, dA, lddA, dB, lddB, dW, dwork, lwork, dinfo));
  CUDA_CALL(hipDeviceSynchronize());
  (void)fprintf(stdout, "%lld,", stopwatch_lap(sw));
  (void)fflush(stdout);
  CUDA_CALL(hipFree(dwork));
  double *const hW = allocHostVec<double>(n + 1u);
  SYSP_CALL(hW);
  CUDA_CALL(hipMemcpy(hW, dW, (n + 1u) * sizeof(double), hipMemcpyDeviceToHost));
  CUDA_CALL(hipDeviceSynchronize());
  CUDA_CALL(hipFree(dW));
  CUDA_CALL(hipMemcpy2D(hG, ldhG * sizeof(hipDoubleComplex), dB, lddB * sizeof(hipDoubleComplex), n * sizeof(hipDoubleComplex), n, hipMemcpyDeviceToHost));
  (void)fprintf(stdout, "%d\n", *reinterpret_cast<const int*>(hW + n));
  (void)fflush(stdout);
  FILE *const l = fopen(strcat(strcpy(fn, argv[4]), ".L"), "wb");
  SYSP_CALL(l);
  SYSI_CALL(fwrite(hW, sizeof(double), n, l) != n);
  SYSI_CALL(fclose(l));
  CUDA_CALL(hipDeviceSynchronize());
  CUDA_CALL(hipFree(dB));
  CUDA_CALL(hipMemcpy2D(hF, ldhF * sizeof(hipDoubleComplex), dA, lddA * sizeof(hipDoubleComplex), n * sizeof(hipDoubleComplex), n, hipMemcpyDeviceToHost));
  FILE *const b = fopen(strcat(strcpy(fn, argv[4]), ".B"), "wb");
  SYSP_CALL(b);
  n2 = n * static_cast<size_t>(n);
  SYSI_CALL(fresize(b, n2 * sizeof(hipDoubleComplex)));
  SYSI_CALL(fwrite_bycol(b, n, n, hG, ldhG));
  SYSI_CALL(fclose(b));
  CUDA_CALL(hipDeviceSynchronize());
  CUDA_CALL(hipFree(dA));
  FILE *const e = fopen(strcat(strcpy(fn, argv[4]), ".E"), "wb");
  SYSP_CALL(e);
  SYSI_CALL(fresize(e, n2 * sizeof(hipDoubleComplex)));
  SYSI_CALL(fwrite_bycol(e, n, n, hF, ldhF));
  SYSI_CALL(fclose(e));
  CUDA_CALL(hipHostFree(hW));
  CUDA_CALL(hipHostFree(hG));
  CUDA_CALL(hipHostFree(hF));
  free(fn);
  freeGPU(cbh, csh);
  return EXIT_SUCCESS;
}
