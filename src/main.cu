// main.cu: test driver.

#include "HZ.hpp"
#include "HZ_L2.hpp"

#include "cuda_memory_helper.hpp"
#include "hdf5_helper.hpp"
#include "my_utils.hpp"

int main(int argc, char *argv[])
{
  if ((8 > argc) || (9 < argc)) {
    (void)fprintf(stderr, "%s DEV SDY SNP0 SNP1 ALG H5F H5G [H5R]\n", argv[0]);
    return EXIT_FAILURE;
  }

  const char *const ca_exe = argv[0];
  const char *const ca_dev = argv[1];
  const char *const ca_sdy = argv[2];
  const char *const ca_snp0 = argv[3];
  const char *const ca_snp1 = argv[4];
  const char *const ca_alg = argv[5];
  const char *const ca_h5f = argv[6];
  const char *const ca_h5g = argv[7];
  const char *const ca_h5r = ((9 == argc) ? argv[8] : static_cast<const char*>(NULL));

  unsigned idadim[HZ_IDADIM_SIZE] = { 0u };
  hid_t fid = static_cast<hid_t>(-1), gid = static_cast<hid_t>(-1);
  size_t ldA = static_cast<size_t>(0u), m = static_cast<size_t>(0u), n = static_cast<size_t>(0u);

  const unsigned n0 = (HZ_L1_NCOLB << 1u);
  const unsigned n1 = (static_cast<unsigned>(atoi(ca_h5g)) + HZ_L1_NCOLB - 1u) / HZ_L1_NCOLB;
  init_strats(ca_sdy, ca_snp0, n0, ca_snp1, n1);

  const int dev = atoi(ca_dev);
  const int dcc = configureGPU(dev);
  if (dcc < 30) {
    (void)snprintf(err_msg, err_msg_size, "Device %d has CC %d < 30", dev, dcc);
    DIE(err_msg);
  }

  const unsigned routine = static_cast<unsigned>(atoi(ca_alg));

  if (!fexist(ca_h5f)) {
    (void)snprintf(err_msg, err_msg_size, "non-existent H5F(%s)", ca_h5f);
    DIE(err_msg);
  }

  HDF5_CALL(H5open());

  HDF5_CALL((fid = H5Fopen(ca_h5f, H5F_ACC_RDONLY, H5P_DEFAULT)));
  HDF5_CALL((gid = H5Gopen2(fid, ca_h5g, H5P_DEFAULT)));

  HDF5_CALL(H5LTread_dataset_int(gid, "IDADIM", reinterpret_cast<int*>(idadim)));

  unsigned
    ldhF = idadim[0],
    ldhG = idadim[0],
    ldhV = idadim[0],
    nrow = idadim[0],
    ncol = idadim[0];

  m = static_cast<size_t>(nrow);
  n = static_cast<size_t>(ncol);

  ldA = static_cast<size_t>(ldhF);
  double *const hF = allocHostMtx<double>(ldA, m, n, true);
  SYSP_CALL(hF);
  ldhF = static_cast<unsigned>(ldA);

  HDF5_CALL(H5LTread_dataset_double(gid, "F", reinterpret_cast<double*>(hF)));

  ldA = static_cast<size_t>(ldhG);
  double *const hG = allocHostMtx<double>(ldA, m, n, true);
  SYSP_CALL(hG);
  ldhG = static_cast<unsigned>(ldA);

  HDF5_CALL(H5LTread_dataset_double(gid, "G", reinterpret_cast<double*>(hG)));

  HDF5_CALL(H5Gclose(gid));
  HDF5_CALL(H5Fclose(fid));

  double *hV = static_cast<double*>(NULL);
  ldA = static_cast<size_t>(ldhV);
  hV = allocHostMtx<double>(ldA, n, n, true);
  SYSP_CALL(hV);
  ldhV = static_cast<unsigned>(ldA);

  double *const hS = allocHostVec<double>(n);
  SYSP_CALL(hS);
  double *const hH = allocHostVec<double>(n);
  SYSP_CALL(hH);
  double *const hK = allocHostVec<double>(n);
  SYSP_CALL(hK);

  unsigned glbSwp = 0u;
  unsigned Long glb_s = MkLong(0u), glb_b = MkLong(0u);
  double timing[4] = { -0.0, -0.0, -0.0, -0.0 };
  int ret = HZ_L2(routine, nrow, ncol, hF, ldhF, hG, ldhG, hV, ldhV, hS, hH, hK, &glbSwp, &glb_s, &glb_b, timing);

  if (ret)
    (void)fprintf(stderr, "%s: error %d\n", ca_exe, ret);
  else {
    (void)fprintf(stdout, "GLB_ROT_S(%20" FmtLong "u), GLB_ROT_B(%20" FmtLong "u)\n", glb_s, glb_b);
    (void)fflush(stdout);
    (void)fprintf(stdout, "%#12.6f s %2u sweeps\n", *timing, glbSwp);
    (void)fflush(stdout);
  }

  if (ca_h5r) {
    HDF5_CALL(fid = fexist(ca_h5r) ?
              H5Fopen(ca_h5r, H5F_ACC_RDWR, H5P_DEFAULT) :
              H5Fcreate(ca_h5r, H5F_ACC_EXCL, H5P_DEFAULT, H5P_DEFAULT));
    HDF5_CALL(gid = H5Gcreate2(fid, ca_h5g, H5P_DEFAULT, H5P_DEFAULT, H5P_DEFAULT));

    idadim[0] = ((ldhF <= ldhG) ? ldhF : ldhG);
    hsize_t dims[2] = { static_cast<hsize_t>(HZ_IDADIM_SIZE), static_cast<hsize_t>(1u) };
    HDF5_CALL(H5LTmake_dataset_int(gid, "IDADIM", 1, dims, reinterpret_cast<int*>(idadim)));

    int istats[4] = { static_cast<int>(CVG), static_cast<int>(STRAT0), static_cast<int>(STRAT1), static_cast<int>(routine) };
    istats[0] = ((ret < 0) ? ret : static_cast<int>(glbSwp));
    dims[0] = static_cast<hsize_t>(4u);
    dims[1] = static_cast<hsize_t>(1u);
    HDF5_CALL(H5LTmake_dataset_int(gid, "ISTATS", 1, dims, istats));

    dims[0] = static_cast<hsize_t>(4u);
    dims[1] = static_cast<hsize_t>(1u);
    HDF5_CALL(H5LTmake_dataset_double(gid, "TIMING", 1, dims, timing));

    if (hF) {
      dims[0] = static_cast<hsize_t>(ncol);
      dims[1] = static_cast<hsize_t>(ldhF);
      HDF5_CALL(H5LTmake_dataset_double(gid, "F", 2, dims, hF));
    }

    if (hG) {
      dims[0] = static_cast<hsize_t>(ncol);
      dims[1] = static_cast<hsize_t>(ldhG);
      HDF5_CALL(H5LTmake_dataset_double(gid, "G", 2, dims, hG));
    }

    if (hV) {
      dims[0] = static_cast<hsize_t>(ncol);
      dims[1] = static_cast<hsize_t>(ldhV);
      HDF5_CALL(H5LTmake_dataset_double(gid, "V", 2, dims, hV));
    }

    if (hS) {
      dims[0] = static_cast<hsize_t>(ncol);
      dims[1] = static_cast<hsize_t>(1u);
      HDF5_CALL(H5LTmake_dataset_double(gid, "SIGMA", 1, dims, hS));
    }

    if (hH) {
      dims[0] = static_cast<hsize_t>(ncol);
      dims[1] = static_cast<hsize_t>(1u);
      HDF5_CALL(H5LTmake_dataset_double(gid, "H", 1, dims, hH));
    }

    if (hK) {
      dims[0] = static_cast<hsize_t>(ncol);
      dims[1] = static_cast<hsize_t>(1u);
      HDF5_CALL(H5LTmake_dataset_double(gid, "K", 1, dims, hK));
    }

    HDF5_CALL(H5Gclose(gid));
    HDF5_CALL(H5Fclose(fid));
  }

  HDF5_CALL(H5close());

  if (hK)
    CUDA_CALL(hipHostFree(hK));
  if (hH)
    CUDA_CALL(hipHostFree(hH));
  if (hS)
    CUDA_CALL(hipHostFree(hS));
  if (hV)
    CUDA_CALL(hipHostFree(hV));
  if (hG)
    CUDA_CALL(hipHostFree(hG));
  if (hF)
    CUDA_CALL(hipHostFree(hF));

  // for profiling
  CUDA_CALL(hipDeviceSynchronize());
  CUDA_CALL(hipDeviceReset());

  return ret;
}
