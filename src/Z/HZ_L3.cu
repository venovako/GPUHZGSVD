#include "hip/hip_runtime.h"
#include "HZ_L3.hpp"

#include "cuda_memory_helper.hpp"
#include "HZ_L.hpp"
#include "HZ_L2.hpp"

int HZ_L3
(const unsigned routine,    // IN, routine ID, <= 15, (Bb__)_2,
 // bits B, b: block-oriented (else, full-block), level 1 and 2;
 const size_t gpu,          // IN, GPU ID (0 <= gpu < gpus);
 const size_t gpus,         // IN, number of GPUs;
 const size_t mF,           // IN, number of rows of F, == 0 (mod 64);
 const size_t mG,           // IN, number of rows of G, == 0 (mod 64);
 const size_t n,            // IN, number of columns, <= min(mF, mG), == 0 (mod 32);
 const size_t n_gpu,        // IN, number of columns per GPU (2 * n_col);
 const size_t n_col,        // IN, number of columns in a block column;
 cuD *const hFD,            // INOUT, ldhF x n_gpu host array in Fortran order;
 cuJ *const hFJ,            // INOUT, ldhF x n_gpu host array in Fortran order;
 const size_t ldhF,         // IN, leading dimension of F, >= mF;
 cuD *const hGD,            // INOUT, ldhG x n_gpu host array in Fortran order;
 cuJ *const hGJ,            // INOUT, ldhG x n_gpu host array in Fortran order;
 const size_t ldhG,         // IN, leading dimension of G, >= mG;
 cuD *const hVD,            // OUT, ldhV x n_gpu host array in Fortran order;
 cuJ *const hVJ,            // OUT, ldhV x n_gpu host array in Fortran order;
 const size_t ldhV,         // IN, leading dimension of V, >= n;
 double *const hS,          // OUT, the generalized singular values, optionally sorted in descending order;
 double *const hH,          // ||F_i||_2/sqrt(||F_i||_2^2 + ||G_i||_2^2);
 double *const hK,          // ||G_i||_2/sqrt(||F_i||_2^2 + ||G_i||_2^2);
 unsigned &glbSwp,          // OUT, number of sweeps at the outermost level;
 unsigned long long &glb_s, // OUT, number of rotations;
 unsigned long long &glb_b, // OUT, number of ``big'' rotations;
 double *const timing       // OUT, optional, in seconds, double[4];
) throw()
{
  switch (routine) {
  case 12:
  case 8u:
  case 4u:
  case 0u:
    break;
  default:
    return -1;
  }

  if (gpu >= gpus)
    return -2;
  if (!gpus)
    return -3;

  if (!mF)
    return -4;
  if (!mG)
    return -5;
  if (!n)
    return -6;
  if (!n_gpu)
    return -7;
  if (!n_col)
    return -8;

  if (!hFD)
    return -9;
  if (!hFJ)
    return -10;
  if (ldhF < mF)
    return -11;

  if (!hGD)
    return -12;
  if (!hGJ)
    return -13;
  if (ldhG < mG)
    return -14;

  if (!hVD)
    return -15;
  if (!hVJ)
    return -16;
  if (ldhV < n)
    return -17;

  if (!hS)
    return -18;
  if (!hH)
    return -19;
  if (!hK)
    return -20;

  size_t lddF = mF;
  cuD *const dFD = allocDeviceMtx<cuD>(lddF, mF, n_gpu, true);
  cuJ *const dFJ = allocDeviceMtx<cuJ>(lddF, mF, n_gpu, true);

  size_t lddG = mG;
  cuD *const dGD = allocDeviceMtx<cuD>(lddG, mG, n_gpu, true);
  cuJ *const dGJ = allocDeviceMtx<cuJ>(lddG, mG, n_gpu, true);

  size_t lddV = n;
  cuD *const dVD = allocDeviceMtx<cuD>(lddV, n, n_gpu, true);
  cuJ *const dVJ = allocDeviceMtx<cuJ>(lddV, n, n_gpu, true);

  double *const dS = allocDeviceVec<double>(n_gpu);
  double *const dH = allocDeviceVec<double>(n_gpu);
  double *const dK = allocDeviceVec<double>(n_gpu);

  const unsigned swp = HZ_NSWEEP;
  unsigned alg = (routine | 1u);

  CUDA_CALL(hipMemcpy2DAsync(dFD, lddF * sizeof(cuD), hFD, ldhF * sizeof(double), mF * sizeof(cuD), n_gpu, hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy2DAsync(dFJ, lddF * sizeof(cuJ), hFJ, ldhF * sizeof(double), mF * sizeof(cuJ), n_gpu, hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy2DAsync(dGD, lddG * sizeof(cuD), hGD, ldhG * sizeof(double), mG * sizeof(cuD), n_gpu, hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy2DAsync(dGJ, lddG * sizeof(cuJ), hGJ, ldhG * sizeof(double), mG * sizeof(cuJ), n_gpu, hipMemcpyHostToDevice));
  CUDA_CALL(hipMemset2DAsync(dVD, lddV * sizeof(cuD), 0, n * sizeof(cuD), n_gpu));
  CUDA_CALL(hipMemset2DAsync(dVJ, lddV * sizeof(cuJ), 0, n * sizeof(cuJ), n_gpu));
  CUDA_CALL(hipMemsetAsync(dS, 0, n_gpu * sizeof(double)));
  CUDA_CALL(hipMemsetAsync(dH, 0, n_gpu * sizeof(double)));
  CUDA_CALL(hipMemsetAsync(dK, 0, n_gpu * sizeof(double)));
  CUDA_CALL(hipDeviceSynchronize());

  CUDA_CALL(hipMemcpy2DAsync(hFD, ldhF * sizeof(double), dFD, lddF * sizeof(cuD), mF * sizeof(cuD), n_gpu, hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy2DAsync(hFJ, ldhF * sizeof(double), dFJ, lddF * sizeof(cuJ), mF * sizeof(cuJ), n_gpu, hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy2DAsync(hGD, ldhG * sizeof(double), dGD, lddG * sizeof(cuD), mG * sizeof(cuD), n_gpu, hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy2DAsync(hGJ, ldhG * sizeof(double), dGJ, lddG * sizeof(cuJ), mG * sizeof(cuJ), n_gpu, hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy2DAsync(hVD, ldhV * sizeof(double), dVD, lddV * sizeof(cuD), n * sizeof(cuD), n_gpu, hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy2DAsync(hVJ, ldhV * sizeof(double), dVJ, lddV * sizeof(cuJ), n * sizeof(cuJ), n_gpu, hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpyAsync(hS, dS, n_gpu * sizeof(double), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpyAsync(hH, dH, n_gpu * sizeof(double), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpyAsync(hK, dK, n_gpu * sizeof(double), hipMemcpyDeviceToHost));
  CUDA_CALL(hipDeviceSynchronize());

  CUDA_CALL(hipFree(static_cast<void*>(dK)));
  CUDA_CALL(hipFree(static_cast<void*>(dH)));
  CUDA_CALL(hipFree(static_cast<void*>(dS)));
  CUDA_CALL(hipFree(static_cast<void*>(dVJ)));
  CUDA_CALL(hipFree(static_cast<void*>(dVD)));
  CUDA_CALL(hipFree(static_cast<void*>(dGJ)));
  CUDA_CALL(hipFree(static_cast<void*>(dGD)));
  CUDA_CALL(hipFree(static_cast<void*>(dFJ)));
  CUDA_CALL(hipFree(static_cast<void*>(dFD)));
  CUDA_CALL(hipDeviceSynchronize());

  return 0;
}
