#include "hip/hip_runtime.h"
#include "HZ_L3.hpp"

#include "HZ_L.hpp"
#include "HZ_L2.hpp"
#include "cuda_memory_helper.hpp"
#include "device_code.hpp"

int HZ_L3
(const unsigned routine,    // IN, routine ID, <= 15, (BbN_)_2,
 // bits B, b: block-oriented (else, full-block), level 1 and 2, N: no sort;
 const size_t gpu,          // IN, GPU ID (0 <= gpu < gpus);
 const size_t gpus,         // IN, number of GPUs;
 const size_t mF,           // IN, number of rows of F, == 0 (mod 64);
 const size_t mG,           // IN, number of rows of G, == 0 (mod 64);
 const size_t n,            // IN, number of columns, <= min(mF, mG), == 0 (mod 32);
 const size_t n_gpu,        // IN, number of columns per GPU (2 * n_col);
 const size_t n_col,        // IN, number of columns in a block column;
 cuD *const hFD,            // INOUT, ldhF x n_gpu host array in Fortran order;
 cuJ *const hFJ,            // INOUT, ldhF x n_gpu host array in Fortran order;
 const size_t ldhF,         // IN, leading dimension of F, >= mF;
 cuD *const hGD,            // INOUT, ldhG x n_gpu host array in Fortran order;
 cuJ *const hGJ,            // INOUT, ldhG x n_gpu host array in Fortran order;
 const size_t ldhG,         // IN, leading dimension of G, >= mG;
 cuD *const hVD,            // OUT, ldhV x n_gpu host array in Fortran order;
 cuJ *const hVJ,            // OUT, ldhV x n_gpu host array in Fortran order;
 const size_t ldhV,         // IN, leading dimension of V, >= n;
 double *const hS,          // OUT, the generalized singular values, optionally sorted in descending order;
 double *const hH,          // ||F_i||_2/sqrt(||F_i||_2^2 + ||G_i||_2^2);
 double *const hK,          // ||G_i||_2/sqrt(||F_i||_2^2 + ||G_i||_2^2);
 unsigned &glbSwp,          // OUT, number of sweeps at the outermost level;
 unsigned long long &glb_s, // OUT, number of rotations;
 unsigned long long &glb_b, // OUT, number of ``big'' rotations;
 double &timing             // OUT, in seconds;
) throw()
{
  if (routine >= 16)
    return -1;

  if (gpu >= gpus)
    return -2;
  if (!gpus)
    return -3;

  if (!mF)
    return -4;
  if (!mG)
    return -5;
  if (!n)
    return -6;
  if (!n_gpu)
    return -7;
  if (!n_col)
    return -8;

  if (!hFD)
    return -9;
  if (!hFJ)
    return -10;
  if (ldhF < mF)
    return -11;

  if (!hGD)
    return -12;
  if (!hGJ)
    return -13;
  if (ldhG < mG)
    return -14;

  if (!hVD)
    return -15;
  if (!hVJ)
    return -16;
  if (ldhV < n)
    return -17;

  if (!hS)
    return -18;
  if (!hH)
    return -19;
  if (!hK)
    return -20;

  size_t lddF = mF;
  cuD *const dFD = allocDeviceMtx<cuD>(lddF, mF, n_gpu, true);
  cuJ *const dFJ = allocDeviceMtx<cuJ>(lddF, mF, n_gpu, true);
  if (lddF != ldhF) {
    DIE("lddF != ldhF");
  }
  
  size_t lddG = mG;
  cuD *const dGD = allocDeviceMtx<cuD>(lddG, mG, n_gpu, true);
  cuJ *const dGJ = allocDeviceMtx<cuJ>(lddG, mG, n_gpu, true);
  if (lddG != ldhG) {
    DIE("lddG != ldhG");
  }

  size_t lddV = n;
  cuD *const dVD = allocDeviceMtx<cuD>(lddV, n, n_gpu, true);
  cuJ *const dVJ = allocDeviceMtx<cuJ>(lddV, n, n_gpu, true);
  if (lddV != ldhV) {
    DIE("lddV != ldhV");
  }

  double *const dS = allocDeviceVec<double>(n_gpu);
  double *const dH = allocDeviceVec<double>(n_gpu);
  double *const dK = allocDeviceVec<double>(n_gpu);

  unsigned long long *const dC = allocDeviceVec<unsigned long long>(static_cast<size_t>(STRAT1_PAIRS) * 2u);
  unsigned long long *const hC = allocHostVec<unsigned long long>(static_cast<size_t>(STRAT1_PAIRS) * 2u);
  CUDA_CALL(hipDeviceSynchronize());

  if (MPI_Barrier(MPI_COMM_WORLD)) {
    DIE("MPI_Barrier(init)");
  }
  long long all_tim = 0ll, swp_tim = 0ll;
  stopwatch_reset(all_tim);
  glb_s = 0ull;
  glb_b = 0ull;
  glbSwp = 0u;
  timing = 0.0;

  initSymbols(dFD,dFJ, dGD,dGJ, dVD,dVJ, dS,dH,dK, dC, mF,mG,n,n_gpu, lddF,lddG,lddV, ((routine & HZ_BO_1) ? 1u : HZ_NSWEEP));
  CUDA_CALL(hipMemcpy2D(dFD, lddF * sizeof(cuD), hFD, ldhF * sizeof(cuD), mF * sizeof(cuD), n_gpu, hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy2D(dFJ, lddF * sizeof(cuJ), hFJ, ldhF * sizeof(cuJ), mF * sizeof(cuJ), n_gpu, hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy2D(dGD, lddG * sizeof(cuD), hGD, ldhG * sizeof(cuD), mG * sizeof(cuD), n_gpu, hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy2D(dGJ, lddG * sizeof(cuJ), hGJ, ldhG * sizeof(cuJ), mG * sizeof(cuJ), n_gpu, hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy2D(dVD, lddV * sizeof(cuD), hVD, ldhV * sizeof(cuD), n * sizeof(cuD), n_gpu, hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy2D(dVJ, lddV * sizeof(cuJ), hVJ, ldhV * sizeof(cuJ), n * sizeof(cuJ), n_gpu, hipMemcpyHostToDevice));
  CUDA_CALL(hipDeviceSynchronize());
  const unsigned p = static_cast<unsigned>(strat2[0u][gpu][0u][0u]);
  const unsigned q = static_cast<unsigned>(strat2[0u][gpu][0u][1u]);
  const size_t ifc0 = p * n_col;
  const size_t ifc1 = q * n_col;
  initV(((CVG == 0) || (CVG == 1) || (CVG == 4) || (CVG == 5)), n_gpu, ifc0, ifc1);
  CUDA_CALL(hipDeviceSynchronize());

  stopwatch_reset(swp_tim);

  while (glbSwp < HZ_NSWEEP) {
    unsigned swp_swp = 0u;
    unsigned long long swp_rot[2u] = { 0ull, 0ull };
    if (!gpu) {
      (void)fprintf(stdout, "%2u: ", glbSwp);
      (void)fflush(stdout);
    }
    for (unsigned stp = 0u; stp < STRAT2_STEPS; ++stp) {
#ifndef NDEBUG
      if (!gpu) {
        (void)fprintf(stdout, "%u", stp);
        (void)fflush(stdout);
      }
#endif // !NDEBUG
      // p = static_cast<unsigned>(strat2[stp][gpu][0u][0u]);
      // q = static_cast<unsigned>(strat2[stp][gpu][0u][1u]);

      int sp = static_cast<int>(strat2[stp][gpu][1u][0u]);
      const int tp = (sp ? ((sp < 0) ? 0 : 6) : -1);
      if (tp == -1) { DIE("tp"); }
      sp = abs(sp) - 1;

      int sq = static_cast<int>(strat2[stp][gpu][1u][1u]);
      const int tq = (sq ? ((sq < 0) ? 0 : 6) : -1);
      if (tq == -1) { DIE("tq"); }
      sq = abs(sq) - 1;

      MPI_Request r[24u] =
        { MPI_REQUEST_NULL, MPI_REQUEST_NULL, MPI_REQUEST_NULL, MPI_REQUEST_NULL, MPI_REQUEST_NULL, MPI_REQUEST_NULL,
          MPI_REQUEST_NULL, MPI_REQUEST_NULL, MPI_REQUEST_NULL, MPI_REQUEST_NULL, MPI_REQUEST_NULL, MPI_REQUEST_NULL,
          MPI_REQUEST_NULL, MPI_REQUEST_NULL, MPI_REQUEST_NULL, MPI_REQUEST_NULL, MPI_REQUEST_NULL, MPI_REQUEST_NULL,
          MPI_REQUEST_NULL, MPI_REQUEST_NULL, MPI_REQUEST_NULL, MPI_REQUEST_NULL, MPI_REQUEST_NULL, MPI_REQUEST_NULL };

      MPI_Status s[24u];
      (void)memset(s, 0, sizeof(s));

      if (MPI_Irecv(hFD, (ldhF * n_col), MPI_DOUBLE, MPI_ANY_SOURCE, 1, MPI_COMM_WORLD, (r + 0u))) {
        DIE("MPI_Irecv(FD)p");
      }
      if (MPI_Irecv(hFJ, (ldhF * n_col), MPI_DOUBLE, MPI_ANY_SOURCE, 2, MPI_COMM_WORLD, (r + 1u))) {
        DIE("MPI_Irecv(FJ)p");
      }
      if (MPI_Irecv(hGD, (ldhG * n_col), MPI_DOUBLE, MPI_ANY_SOURCE, 3, MPI_COMM_WORLD, (r + 2u))) {
        DIE("MPI_Irecv(GD)p");
      }
      if (MPI_Irecv(hGJ, (ldhG * n_col), MPI_DOUBLE, MPI_ANY_SOURCE, 4, MPI_COMM_WORLD, (r + 3u))) {
        DIE("MPI_Irecv(GJ)p");
      }
      if (MPI_Irecv(hVD, (ldhV * n_col), MPI_DOUBLE, MPI_ANY_SOURCE, 5, MPI_COMM_WORLD, (r + 4u))) {
        DIE("MPI_Irecv(VD)p");
      }
      if (MPI_Irecv(hVJ, (ldhV * n_col), MPI_DOUBLE, MPI_ANY_SOURCE, 6, MPI_COMM_WORLD, (r + 5u))) {
        DIE("MPI_Irecv(VJ)p");
      }

      if (MPI_Irecv((hFD + ldhF * n_col), (ldhF * n_col), MPI_DOUBLE, MPI_ANY_SOURCE, 7, MPI_COMM_WORLD, (r + 6u))) {
        DIE("MPI_Irecv(FD)q");
      }
      if (MPI_Irecv((hFJ + ldhF * n_col), (ldhF * n_col), MPI_DOUBLE, MPI_ANY_SOURCE, 8, MPI_COMM_WORLD, (r + 7u))) {
        DIE("MPI_Irecv(FJ)q");
      }
      if (MPI_Irecv((hGD + ldhG * n_col), (ldhG * n_col), MPI_DOUBLE, MPI_ANY_SOURCE, 9, MPI_COMM_WORLD, (r + 8u))) {
        DIE("MPI_Irecv(GD)q");
      }
      if (MPI_Irecv((hGJ + ldhG * n_col), (ldhG * n_col), MPI_DOUBLE, MPI_ANY_SOURCE, 10, MPI_COMM_WORLD, (r + 9u))) {
        DIE("MPI_Irecv(GJ)q");
      }
      if (MPI_Irecv((hVD + ldhV * n_col), (ldhV * n_col), MPI_DOUBLE, MPI_ANY_SOURCE, 11, MPI_COMM_WORLD, (r + 10u))) {
        DIE("MPI_Irecv(VD)q");
      }
      if (MPI_Irecv((hVJ + ldhV * n_col), (ldhV * n_col), MPI_DOUBLE, MPI_ANY_SOURCE, 12, MPI_COMM_WORLD, (r + 11u))) {
        DIE("MPI_Irecv(VJ)q");
      }

      unsigned swp2 = 0u;
      unsigned long long rot2s = 0ull, rot2b = 0ull;
      const int ret = HZ_L2_gpu(routine, mF,mG,n_gpu, dFD,dFJ,lddF, dGD,dGJ,lddG, dVD,dVJ,lddV, dS,dH,dK, hC,dC, swp2,rot2s,rot2b);
      if (ret) {
        (void)snprintf(err_msg, err_msg_size, "HZ_L2_gpu @GPU(%u) SWP(%u) STP(%u): %d", gpu, glbSwp, stp, ret);
        DIE(err_msg);
      }
      if (swp2 > swp_swp)
        swp_swp = swp2;
      swp_rot[0u] += rot2s;
      swp_rot[1u] += rot2b;

      if (MPI_Isend(dFD, (ldhF * n_col), MPI_DOUBLE, sp, (1 + tp), MPI_COMM_WORLD, (r + 12u))) {
        DIE("MPI_Isend(FD)p");
      }
      if (MPI_Isend(dFJ, (ldhF * n_col), MPI_DOUBLE, sp, (2 + tp), MPI_COMM_WORLD, (r + 13u))) {
        DIE("MPI_Isend(FJ)p");
      }
      if (MPI_Isend(dGD, (ldhG * n_col), MPI_DOUBLE, sp, (3 + tp), MPI_COMM_WORLD, (r + 14u))) {
        DIE("MPI_Isend(GD)p");
      }
      if (MPI_Isend(dGJ, (ldhG * n_col), MPI_DOUBLE, sp, (4 + tp), MPI_COMM_WORLD, (r + 15u))) {
        DIE("MPI_Isend(GJ)p");
      }
      if (MPI_Isend(dVD, (ldhV * n_col), MPI_DOUBLE, sp, (5 + tp), MPI_COMM_WORLD, (r + 16u))) {
        DIE("MPI_Isend(VD)p");
      }
      if (MPI_Isend(dVJ, (ldhV * n_col), MPI_DOUBLE, sp, (6 + tp), MPI_COMM_WORLD, (r + 17u))) {
        DIE("MPI_Isend(VJ)p");
      }

      if (MPI_Isend((dFD + ldhF * n_col), (ldhF * n_col), MPI_DOUBLE, sq, (1 + tq), MPI_COMM_WORLD, (r + 18u))) {
        DIE("MPI_Isend(FD)q");
      }
      if (MPI_Isend((dFJ + ldhF * n_col), (ldhF * n_col), MPI_DOUBLE, sq, (2 + tq), MPI_COMM_WORLD, (r + 19u))) {
        DIE("MPI_Isend(FJ)q");
      }
      if (MPI_Isend((dGD + ldhG * n_col), (ldhG * n_col), MPI_DOUBLE, sq, (3 + tq), MPI_COMM_WORLD, (r + 20u))) {
        DIE("MPI_Isend(GD)q");
      }
      if (MPI_Isend((dGJ + ldhG * n_col), (ldhG * n_col), MPI_DOUBLE, sq, (4 + tq), MPI_COMM_WORLD, (r + 21u))) {
        DIE("MPI_Isend(GJ)q");
      }
      if (MPI_Isend((dVD + ldhV * n_col), (ldhV * n_col), MPI_DOUBLE, sq, (5 + tq), MPI_COMM_WORLD, (r + 22u))) {
        DIE("MPI_Isend(VD)q");
      }
      if (MPI_Isend((dVJ + ldhV * n_col), (ldhV * n_col), MPI_DOUBLE, sq, (6 + tq), MPI_COMM_WORLD, (r + 23u))) {
        DIE("MPI_Isend(VJ)q");
      }

      if (MPI_Waitall(24, r, s)) {
        DIE("MPI_Waitall");
      }
      for (unsigned i = 0u; i < 24u; ++i) {
        if (s[i].MPI_ERROR) {
          DIE("MPI_Status");
        }
      }
      CUDA_CALL(hipMemcpy2D(dFD, lddF * sizeof(cuD), hFD, ldhF * sizeof(cuD), mF * sizeof(cuD), n_gpu, hipMemcpyHostToDevice));
      CUDA_CALL(hipMemcpy2D(dFJ, lddF * sizeof(cuJ), hFJ, ldhF * sizeof(cuJ), mF * sizeof(cuJ), n_gpu, hipMemcpyHostToDevice));
      CUDA_CALL(hipMemcpy2D(dGD, lddG * sizeof(cuD), hGD, ldhG * sizeof(cuD), mG * sizeof(cuD), n_gpu, hipMemcpyHostToDevice));
      CUDA_CALL(hipMemcpy2D(dGJ, lddG * sizeof(cuJ), hGJ, ldhG * sizeof(cuJ), mG * sizeof(cuJ), n_gpu, hipMemcpyHostToDevice));
      CUDA_CALL(hipMemcpy2D(dVD, lddV * sizeof(cuD), hVD, ldhV * sizeof(cuD), n * sizeof(cuD), n_gpu, hipMemcpyHostToDevice));
      CUDA_CALL(hipMemcpy2D(dVJ, lddV * sizeof(cuJ), hVJ, ldhV * sizeof(cuJ), n * sizeof(cuJ), n_gpu, hipMemcpyHostToDevice));
      CUDA_CALL(hipDeviceSynchronize());
#ifndef NDEBUG
      if (!gpu) {
        (void)fprintf(stdout, ";");
        (void)fflush(stdout);
      }
#endif // !NDEBUG
      if (MPI_Barrier(MPI_COMM_WORLD)) {
        DIE("MPI_Barrier");
      }
    }
    unsigned max_swp = 0u;
    if (MPI_Allreduce(&swp_swp, &max_swp, 1, MPI_UNSIGNED, MPI_MAX, MPI_COMM_WORLD)) {
      DIE("MPI_Allreduce(max_swp)");
    }
    unsigned long long all_rot[2u] = { 0ull, 0ull };
    if (MPI_Allreduce(swp_rot, all_rot, 2, MPI_UNSIGNED_LONG_LONG, MPI_SUM, MPI_COMM_WORLD)) {
      DIE("MPI_Allreduce(all_rot)");
    }
    glb_s += all_rot[0u];
    glb_b += all_rot[1u];
    ++glbSwp;

    if (!gpu) {
      (void)fprintf(stdout, "MAX2SWP(%2u), ROT_S(%13llu), ROT_B(%13llu), TIME(%#14.6f s)\n", max_swp, all_rot[0u], all_rot[1u], (stopwatch_lap(swp_tim) * TS2S));
      (void)fflush(stdout);
    }
    if (!all_rot[1u])
      break;
  }

  initS(1, n_gpu);
  CUDA_CALL(hipDeviceSynchronize());

  CUDA_CALL(hipMemcpy2D(hFD, ldhF * sizeof(cuD), dFD, lddF * sizeof(cuD), mF * sizeof(cuD), n_gpu, hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy2D(hFJ, ldhF * sizeof(cuJ), dFJ, lddF * sizeof(cuJ), mF * sizeof(cuJ), n_gpu, hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy2D(hGD, ldhG * sizeof(cuD), dGD, lddG * sizeof(cuD), mG * sizeof(cuD), n_gpu, hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy2D(hGJ, ldhG * sizeof(cuJ), dGJ, lddG * sizeof(cuJ), mG * sizeof(cuJ), n_gpu, hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy2D(hVD, ldhV * sizeof(cuD), dVD, lddV * sizeof(cuD), n * sizeof(cuD), n_gpu, hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy2D(hVJ, ldhV * sizeof(cuJ), dVJ, lddV * sizeof(cuJ), n * sizeof(cuJ), n_gpu, hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(hS, dS, n_gpu * sizeof(double), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(hH, dH, n_gpu * sizeof(double), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(hK, dK, n_gpu * sizeof(double), hipMemcpyDeviceToHost));
  CUDA_CALL(hipDeviceSynchronize());

  if (MPI_Barrier(MPI_COMM_WORLD)) {
    DIE("MPI_Barrier(fini)");
  }
  timing = (stopwatch_lap(all_tim) * TS2S);

  CUDA_CALL(hipHostFree(hC));
  CUDA_CALL(hipFree(dC));
  CUDA_CALL(hipFree(dK));
  CUDA_CALL(hipFree(dH));
  CUDA_CALL(hipFree(dS));
  CUDA_CALL(hipFree(dVJ));
  CUDA_CALL(hipFree(dVD));
  CUDA_CALL(hipFree(dGJ));
  CUDA_CALL(hipFree(dGD));
  CUDA_CALL(hipFree(dFJ));
  CUDA_CALL(hipFree(dFD));
  CUDA_CALL(hipDeviceSynchronize());

  return 0;
}
