#include "hip/hip_runtime.h"
#include "HZ_L3.hpp"

#include "HZ_L2.hpp"

int HZ_L3
(const unsigned routine,    // IN, routine ID, <= 15, (Bb__)_2,
 // bits B, b: block-oriented (else, full-block), level 1 and 2;
 const size_t gpu,          // IN, GPU ID (0 <= gpu < gpus);
 const size_t gpus,         // IN, number of GPUs;
 const size_t mF,           // IN, number of rows of F, == 0 (mod 64);
 const size_t mG,           // IN, number of rows of G, == 0 (mod 64);
 const size_t n,            // IN, number of columns, <= min(mF, mG), == 0 (mod 32);
 const size_t n_gpu,        // IN, number of columns per GPU (2 * n_col);
 const size_t n_col,        // IN, number of columns in a block column;
 cuD *const hFD,            // INOUT, ldhF x n_gpu host array in Fortran order;
 cuJ *const hFJ,            // INOUT, ldhF x n_gpu host array in Fortran order;
 const size_t ldhF,         // IN, leading dimension of F, >= mF;
 cuD *const hGD,            // INOUT, ldhG x n_gpu host array in Fortran order;
 cuJ *const hGJ,            // INOUT, ldhG x n_gpu host array in Fortran order;
 const size_t ldhG,         // IN, leading dimension of G, >= mG;
 cuD *const hVD,            // OUT, ldhV x n_gpu host array in Fortran order;
 cuJ *const hVJ,            // OUT, ldhV x n_gpu host array in Fortran order;
 const size_t ldhV,         // IN, leading dimension of V, >= n;
 double *const hS,          // OUT, the generalized singular values, optionally sorted in descending order;
 double *const hH,          // ||F_i||_2/sqrt(||F_i||_2^2 + ||G_i||_2^2);
 double *const hK,          // ||G_i||_2/sqrt(||F_i||_2^2 + ||G_i||_2^2);
 unsigned &glbSwp,          // OUT, number of sweeps at the outermost level;
 unsigned long long &glb_s, // OUT, number of rotations;
 unsigned long long &glb_b, // OUT, number of ``big'' rotations;
 double *const timing       // OUT, optional, in seconds, double[4];
) throw()
{
  return 0;
}
