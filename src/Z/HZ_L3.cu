#include "hip/hip_runtime.h"
#include "HZ_L3.hpp"

#include "HZ_L.hpp"
#include "HZ_L2.hpp"
#include "cuda_memory_helper.hpp"

int HZ_L3
(const unsigned routine,    // IN, routine ID, <= 15, (BbN_)_2,
 // bits B, b: block-oriented (else, full-block), level 1 and 2, N: no sort;
 const size_t gpu,          // IN, GPU ID (0 <= gpu < gpus);
 const size_t gpus,         // IN, number of GPUs;
 const size_t mF,           // IN, number of rows of F, == 0 (mod 64);
 const size_t mG,           // IN, number of rows of G, == 0 (mod 64);
 const size_t n,            // IN, number of columns, <= min(mF, mG), == 0 (mod 32);
 const size_t n_gpu,        // IN, number of columns per GPU (2 * n_col);
 const size_t n_col,        // IN, number of columns in a block column;
 cuD *const hFD,            // INOUT, ldhF x n_gpu host array in Fortran order;
 cuJ *const hFJ,            // INOUT, ldhF x n_gpu host array in Fortran order;
 const size_t ldhF,         // IN, leading dimension of F, >= mF;
 cuD *const hGD,            // INOUT, ldhG x n_gpu host array in Fortran order;
 cuJ *const hGJ,            // INOUT, ldhG x n_gpu host array in Fortran order;
 const size_t ldhG,         // IN, leading dimension of G, >= mG;
 cuD *const hVD,            // OUT, ldhV x n_gpu host array in Fortran order;
 cuJ *const hVJ,            // OUT, ldhV x n_gpu host array in Fortran order;
 const size_t ldhV,         // IN, leading dimension of V, >= n;
 double *const hS,          // OUT, the generalized singular values, optionally sorted in descending order;
 double *const hH,          // ||F_i||_2/sqrt(||F_i||_2^2 + ||G_i||_2^2);
 double *const hK,          // ||G_i||_2/sqrt(||F_i||_2^2 + ||G_i||_2^2);
 unsigned &glbSwp,          // OUT, number of sweeps at the outermost level;
 unsigned long long &glb_s, // OUT, number of rotations;
 unsigned long long &glb_b, // OUT, number of ``big'' rotations;
 double &timing             // OUT, in seconds;
) throw()
{
  if (routine >= 16)
    return -1;

  if (gpu >= gpus)
    return -2;
  if (!gpus)
    return -3;

  if (!mF)
    return -4;
  if (!mG)
    return -5;
  if (!n)
    return -6;
  if (!n_gpu)
    return -7;
  if (!n_col)
    return -8;

  if (!hFD)
    return -9;
  if (!hFJ)
    return -10;
  if (ldhF < mF)
    return -11;

  if (!hGD)
    return -12;
  if (!hGJ)
    return -13;
  if (ldhG < mG)
    return -14;

  if (!hVD)
    return -15;
  if (!hVJ)
    return -16;
  if (ldhV < n)
    return -17;

  if (!hS)
    return -18;
  if (!hH)
    return -19;
  if (!hK)
    return -20;

  size_t lddF = mF;
  cuD *const dFD = allocDeviceMtx<cuD>(lddF, mF, n_gpu, true);
  cuJ *const dFJ = allocDeviceMtx<cuJ>(lddF, mF, n_gpu, true);
  if (lddF != ldhF) {
    DIE("lddF != ldhF");
  }
  
  size_t lddG = mG;
  cuD *const dGD = allocDeviceMtx<cuD>(lddG, mG, n_gpu, true);
  cuJ *const dGJ = allocDeviceMtx<cuJ>(lddG, mG, n_gpu, true);
  if (lddG != ldhG) {
    DIE("lddG != ldhG");
  }

  size_t lddV = n;
  cuD *const dVD = allocDeviceMtx<cuD>(lddV, n, n_gpu, true);
  cuJ *const dVJ = allocDeviceMtx<cuJ>(lddV, n, n_gpu, true);
  if (lddV != ldhV) {
    DIE("lddV != ldhV");
  }

  double *const dS = allocDeviceVec<double>(n_gpu);
  double *const dH = allocDeviceVec<double>(n_gpu);
  double *const dK = allocDeviceVec<double>(n_gpu);

  CUDA_CALL(hipDeviceSynchronize());
  if (MPI_Barrier(MPI_COMM_WORLD)) {
    DIE("MPI_Barrier(init)");
  }
  long long all_tim = 0ll, swp_tim = 0ll;
  stopwatch_reset(all_tim);
  glb_s = 0ull;
  glb_b = 0ull;
  glbSwp = 0u;
  timing = 0.0;
  unsigned alg = (routine | 1u);

  CUDA_CALL(hipMemset2DAsync(dVD, lddV * sizeof(cuD), 0, n * sizeof(cuD), n_gpu));
  CUDA_CALL(hipMemset2DAsync(dVJ, lddV * sizeof(cuJ), 0, n * sizeof(cuJ), n_gpu));
  CUDA_CALL(hipMemsetAsync(dH, 0, n_gpu * sizeof(double)));
  CUDA_CALL(hipMemsetAsync(dK, 0, n_gpu * sizeof(double)));
  CUDA_CALL(hipDeviceSynchronize());

  stopwatch_reset(swp_tim);

  while (glbSwp < HZ_NSWEEP) {
    unsigned swp_swp = 0u;
    unsigned long long swp_rot[2u] = { 0ull, 0ull };
    for (unsigned stp = 0u; stp < STRAT2_STEPS; ++stp) {
      CUDA_CALL(hipMemcpy2DAsync(dFD, lddF * sizeof(cuD), hFD, ldhF * sizeof(double), mF * sizeof(cuD), n_gpu, hipMemcpyHostToDevice));
      CUDA_CALL(hipMemcpy2DAsync(dFJ, lddF * sizeof(cuJ), hFJ, ldhF * sizeof(double), mF * sizeof(cuJ), n_gpu, hipMemcpyHostToDevice));
      CUDA_CALL(hipMemcpy2DAsync(dGD, lddG * sizeof(cuD), hGD, ldhG * sizeof(double), mG * sizeof(cuD), n_gpu, hipMemcpyHostToDevice));
      CUDA_CALL(hipMemcpy2DAsync(dGJ, lddG * sizeof(cuJ), hGJ, ldhG * sizeof(double), mG * sizeof(cuJ), n_gpu, hipMemcpyHostToDevice));
      if (stp || glbSwp) {
        CUDA_CALL(hipMemcpy2DAsync(dVD, lddG * sizeof(cuD), hVD, ldhV * sizeof(double), n * sizeof(cuD), n_gpu, hipMemcpyHostToDevice));
        CUDA_CALL(hipMemcpy2DAsync(dVJ, lddG * sizeof(cuJ), hVJ, ldhV * sizeof(double), n * sizeof(cuJ), n_gpu, hipMemcpyHostToDevice));
      }
      CUDA_CALL(hipDeviceSynchronize());

      int sp = static_cast<int>(strat2[stp][gpu][1u][0u]);
      const int tp = (sp ? ((sp < 0) ? 0 : 6) : -1);
      sp = abs(sp) - 1;

      int sq = static_cast<int>(strat2[stp][gpu][1u][1u]);
      const int tq = (sq ? ((sq < 0) ? 0 : 6) : -1);
      sq = abs(sq) - 1;

      MPI_Request r[24u] =
        { MPI_REQUEST_NULL, MPI_REQUEST_NULL, MPI_REQUEST_NULL, MPI_REQUEST_NULL, MPI_REQUEST_NULL, MPI_REQUEST_NULL,
          MPI_REQUEST_NULL, MPI_REQUEST_NULL, MPI_REQUEST_NULL, MPI_REQUEST_NULL, MPI_REQUEST_NULL, MPI_REQUEST_NULL,
          MPI_REQUEST_NULL, MPI_REQUEST_NULL, MPI_REQUEST_NULL, MPI_REQUEST_NULL, MPI_REQUEST_NULL, MPI_REQUEST_NULL,
          MPI_REQUEST_NULL, MPI_REQUEST_NULL, MPI_REQUEST_NULL, MPI_REQUEST_NULL, MPI_REQUEST_NULL, MPI_REQUEST_NULL };

      if (MPI_Irecv(hFD, (ldhF * n_col), MPI_DOUBLE, MPI_ANY_SOURCE, 1, MPI_COMM_WORLD, (r + 0u))) {
        DIE("MPI_Irecv(FD)p");
      }
      if (MPI_Irecv(hFJ, (ldhF * n_col), MPI_DOUBLE, MPI_ANY_SOURCE, 2, MPI_COMM_WORLD, (r + 1u))) {
        DIE("MPI_Irecv(FJ)p");
      }
      if (MPI_Irecv(hGD, (ldhG * n_col), MPI_DOUBLE, MPI_ANY_SOURCE, 3, MPI_COMM_WORLD, (r + 2u))) {
        DIE("MPI_Irecv(GD)p");
      }
      if (MPI_Irecv(hGJ, (ldhG * n_col), MPI_DOUBLE, MPI_ANY_SOURCE, 4, MPI_COMM_WORLD, (r + 3u))) {
        DIE("MPI_Irecv(GJ)p");
      }
      if (MPI_Irecv(hVD, (ldhV * n_col), MPI_DOUBLE, MPI_ANY_SOURCE, 5, MPI_COMM_WORLD, (r + 4u))) {
        DIE("MPI_Irecv(VD)p");
      }
      if (MPI_Irecv(hVJ, (ldhV * n_col), MPI_DOUBLE, MPI_ANY_SOURCE, 6, MPI_COMM_WORLD, (r + 5u))) {
        DIE("MPI_Irecv(VJ)p");
      }

      if (MPI_Irecv((hFD + ldhF * n_col), (ldhF * n_col), MPI_DOUBLE, MPI_ANY_SOURCE, 7, MPI_COMM_WORLD, (r + 6u))) {
        DIE("MPI_Irecv(FD)q");
      }
      if (MPI_Irecv((hFJ + ldhF * n_col), (ldhF * n_col), MPI_DOUBLE, MPI_ANY_SOURCE, 8, MPI_COMM_WORLD, (r + 7u))) {
        DIE("MPI_Irecv(FJ)q");
      }
      if (MPI_Irecv((hGD + ldhG * n_col), (ldhG * n_col), MPI_DOUBLE, MPI_ANY_SOURCE, 9, MPI_COMM_WORLD, (r + 8u))) {
        DIE("MPI_Irecv(GD)q");
      }
      if (MPI_Irecv((hGJ + ldhG * n_col), (ldhG * n_col), MPI_DOUBLE, MPI_ANY_SOURCE, 10, MPI_COMM_WORLD, (r + 9u))) {
        DIE("MPI_Irecv(GJ)q");
      }
      if (MPI_Irecv((hVD + ldhV * n_col), (ldhV * n_col), MPI_DOUBLE, MPI_ANY_SOURCE, 11, MPI_COMM_WORLD, (r + 10u))) {
        DIE("MPI_Irecv(VD)q");
      }
      if (MPI_Irecv((hVJ + ldhV * n_col), (ldhV * n_col), MPI_DOUBLE, MPI_ANY_SOURCE, 12, MPI_COMM_WORLD, (r + 11u))) {
        DIE("MPI_Irecv(VJ)q");
      }

      const unsigned p = static_cast<unsigned>(strat2[stp][gpu][0u][0u]);
      const unsigned q = static_cast<unsigned>(strat2[stp][gpu][0u][1u]);

      const size_t ifc0 = p * n_col;
      const size_t ifc1 = q * n_col;

      unsigned swp2 = 0u;
      unsigned long long rot2s = 0ull, rot2b = 0ull;
      const int ret = HZ_L2_gpu(alg, mF,mG,n_gpu, ifc0,ifc1, dFD,dFJ,lddF, dGD,dGJ,lddG, dVD,dVJ,lddV, hS,dS,dH,dK, swp2,rot2s,rot2b);
      if (ret) {
        (void)snprintf(err_msg, err_msg_size, "HZ_L2_gpu @GPU(%u) SWP(%u) STP(%u): %d", gpu, glbSwp, stp, ret);
        DIE(err_msg);
      }
      if (swp2 > swp_swp)
        swp_swp = swp2;
      swp_rot[0u] += rot2s;
      swp_rot[1u] += rot2b;
   
      if (MPI_Isend(dFD, (lddF * n_col), MPI_DOUBLE, sp, (1 + tp), MPI_COMM_WORLD, (r + 12u))) {
        DIE("MPI_Isend(FD)p");
      }
      if (MPI_Isend(dFJ, (lddF * n_col), MPI_DOUBLE, sp, (2 + tp), MPI_COMM_WORLD, (r + 13u))) {
        DIE("MPI_Isend(FJ)p");
      }
      if (MPI_Isend(dGD, (lddG * n_col), MPI_DOUBLE, sp, (3 + tp), MPI_COMM_WORLD, (r + 14u))) {
        DIE("MPI_Isend(GD)p");
      }
      if (MPI_Isend(dGJ, (lddG * n_col), MPI_DOUBLE, sp, (4 + tp), MPI_COMM_WORLD, (r + 15u))) {
        DIE("MPI_Isend(GJ)p");
      }
      if (MPI_Isend(dVD, (lddV * n_col), MPI_DOUBLE, sp, (5 + tp), MPI_COMM_WORLD, (r + 16u))) {
        DIE("MPI_Isend(VD)p");
      }
      if (MPI_Isend(dVJ, (lddV * n_col), MPI_DOUBLE, sp, (6 + tp), MPI_COMM_WORLD, (r + 17u))) {
        DIE("MPI_Isend(VJ)p");
      }

      if (MPI_Isend((dFD + lddF * n_col), (lddF * n_col), MPI_DOUBLE, sq, (1 + tq), MPI_COMM_WORLD, (r + 18u))) {
        DIE("MPI_Isend(FD)q");
      }
      if (MPI_Isend((dFJ + lddF * n_col), (lddF * n_col), MPI_DOUBLE, sq, (2 + tq), MPI_COMM_WORLD, (r + 19u))) {
        DIE("MPI_Isend(FJ)q");
      }
      if (MPI_Isend((dGD + lddG * n_col), (lddG * n_col), MPI_DOUBLE, sq, (3 + tq), MPI_COMM_WORLD, (r + 20u))) {
        DIE("MPI_Isend(GD)q");
      }
      if (MPI_Isend((dGJ + lddG * n_col), (lddG * n_col), MPI_DOUBLE, sq, (4 + tq), MPI_COMM_WORLD, (r + 21u))) {
        DIE("MPI_Isend(GJ)q");
      }
      if (MPI_Isend((dVD + lddV * n_col), (lddV * n_col), MPI_DOUBLE, sq, (5 + tq), MPI_COMM_WORLD, (r + 22u))) {
        DIE("MPI_Isend(VD)q");
      }
      if (MPI_Isend((dVJ + lddV * n_col), (lddV * n_col), MPI_DOUBLE, sq, (6 + tq), MPI_COMM_WORLD, (r + 23u))) {
        DIE("MPI_Isend(VJ)q");
      }

      if (MPI_Waitall(24, r, MPI_STATUSES_IGNORE)) {
        DIE("MPI_Waitall");
      }
      CUDA_CALL(hipDeviceSynchronize());
    }
    unsigned max_swp = 0u;
    if (MPI_Allreduce(&swp_swp, &max_swp, 1, MPI_UNSIGNED, MPI_MAX, MPI_COMM_WORLD)) {
      DIE("MPI_Allreduce(max_swp)");
    }
    unsigned long long all_rot[2u] = { 0ull, 0ull };
    if (MPI_Allreduce(swp_rot, all_rot, 2, MPI_UNSIGNED_LONG_LONG, MPI_SUM, MPI_COMM_WORLD)) {
      DIE("MPI_Allreduce(all_rot)");
    }
    glb_s += all_rot[0u];
    glb_b += all_rot[1u];
    ++glbSwp;

    if (!gpu) {
      (void)fprintf(stdout, "MAX2SWP(%2u), ROT_S(%10llu), ROT_B(%10llu), TIME(%#12.6f s)\n", max_swp, all_rot[0u], all_rot[1u], (stopwatch_lap(swp_tim) * TS2S));
      (void)fflush(stdout);
    }
    if (!all_rot[1u])
      break;
  }

  // CUDA_CALL(hipMemcpy2DAsync(hFD, ldhF * sizeof(double), dFD, lddF * sizeof(cuD), mF * sizeof(cuD), n_gpu, hipMemcpyDeviceToHost));
  // CUDA_CALL(hipMemcpy2DAsync(hFJ, ldhF * sizeof(double), dFJ, lddF * sizeof(cuJ), mF * sizeof(cuJ), n_gpu, hipMemcpyDeviceToHost));
  // CUDA_CALL(hipMemcpy2DAsync(hGD, ldhG * sizeof(double), dGD, lddG * sizeof(cuD), mG * sizeof(cuD), n_gpu, hipMemcpyDeviceToHost));
  // CUDA_CALL(hipMemcpy2DAsync(hGJ, ldhG * sizeof(double), dGJ, lddG * sizeof(cuJ), mG * sizeof(cuJ), n_gpu, hipMemcpyDeviceToHost));
  // CUDA_CALL(hipMemcpy2DAsync(hVD, ldhV * sizeof(double), dVD, lddV * sizeof(cuD), n * sizeof(cuD), n_gpu, hipMemcpyDeviceToHost));
  // CUDA_CALL(hipMemcpy2DAsync(hVJ, ldhV * sizeof(double), dVJ, lddV * sizeof(cuJ), n * sizeof(cuJ), n_gpu, hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpyAsync(hS, dS, n_gpu * sizeof(double), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpyAsync(hH, dH, n_gpu * sizeof(double), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpyAsync(hK, dK, n_gpu * sizeof(double), hipMemcpyDeviceToHost));
  CUDA_CALL(hipDeviceSynchronize());

  if (MPI_Barrier(MPI_COMM_WORLD)) {
    DIE("MPI_Barrier(fini)");
  }
  timing = (stopwatch_lap(all_tim) * TS2S);

  CUDA_CALL(hipFree(static_cast<void*>(dK)));
  CUDA_CALL(hipFree(static_cast<void*>(dH)));
  CUDA_CALL(hipFree(static_cast<void*>(dS)));
  CUDA_CALL(hipFree(static_cast<void*>(dVJ)));
  CUDA_CALL(hipFree(static_cast<void*>(dVD)));
  CUDA_CALL(hipFree(static_cast<void*>(dGJ)));
  CUDA_CALL(hipFree(static_cast<void*>(dGD)));
  CUDA_CALL(hipFree(static_cast<void*>(dFJ)));
  CUDA_CALL(hipFree(static_cast<void*>(dFD)));
  CUDA_CALL(hipDeviceSynchronize());

  return 0;
}
