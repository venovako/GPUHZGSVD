#include "device_code.hpp"

#include "HZ.hpp"
#include "HZ_L.hpp"
#include "cuda_helper.hpp"

#include "device_code_common.hpp"
#include "device_code_accumV.hpp"
#if ((CVG == 0) || (CVG == 1))
#include "device_code_cdsort_0.hpp"
#include "device_code_nosort_0.hpp"
#elif ((CVG == 2) || (CVG == 3))
#include "device_code_cdsort_1.hpp"
#include "device_code_nosort_1.hpp"
#elif ((CVG == 4) || (CVG == 5))
#include "device_code_cdsort_2.hpp"
#include "device_code_nosort_2.hpp"
#elif ((CVG == 6) || (CVG == 7))
#include "device_code_cdsort_3.hpp"
#include "device_code_nosort_3.hpp"
#else /* unknown CVG */
#error CVG unknown
#endif /* ?CVG */
#include "device_code_cdsort_accumV.hpp"
#include "device_code_nosort_accumV.hpp"

static const dim3 hzL1bD(HZ_L1_THREADS_PER_BLOCK_X, HZ_L1_THREADS_PER_BLOCK_Y, 1u);

void HZ_L1_sv(const unsigned step, const hipStream_t s) throw()
{
  const dim3 hzL1gD(STRAT1_PAIRS, 1u, 1u);
  const size_t shmD = static_cast<size_t>(0u);
  zHZ_L1_sv<<< hzL1gD, hzL1bD, shmD, s >>>(step);
}

void HZ_L1_v(const unsigned step, const hipStream_t s) throw()
{
  const dim3 hzL1gD(STRAT1_PAIRS, 1u, 1u);
  const size_t shmD = static_cast<size_t>(0u);
  zHZ_L1_v<<< hzL1gD, hzL1bD, shmD, s >>>(step);
}

void initS(const int full, const unsigned nRank, const hipStream_t s) throw()
{
  const dim3 bD(2u * WARP_SZ, 1u, 1u);
  const dim3 gD(udiv_ceil(nRank * WARP_SZ, bD.x), 1u, 1u);
  const size_t shmD = static_cast<size_t>(0u);
  zInitS<<< gD, bD, shmD, s >>>(full);
}

void initV(const int sclV, const unsigned nRank
#ifdef USE_MPI
  , const unsigned ifc0, const unsigned ifc1
#endif /* USE_MPI */
  , const hipStream_t s
) throw()
{
  const dim3 bD(2u * WARP_SZ, 1u, 1u);
  const dim3 gD(udiv_ceil(nRank * WARP_SZ, bD.x), 1u, 1u);
  const size_t shmD = static_cast<size_t>(0u);
  zInitV<<< gD, bD, shmD, s >>>(sclV
#ifdef USE_MPI
    , ifc0, ifc1
#endif /* USE_MPI */
  );
}

void initSymbols
(cuD *const FD, cuJ *const FJ,
 cuD *const GD, cuJ *const GJ,
 cuD *const VD, cuJ *const VJ,
 double *const S,
 double *const H,
 double *const K,
 unsigned long long *const C,
 const unsigned nRowF,
 const unsigned nRowG,
 const unsigned nRowV,
 const unsigned nRank,
 const unsigned ldF,
 const unsigned ldG,
 const unsigned ldV,
 const unsigned nSwp,
 const hipStream_t s
) throw()
{
  const size_t off = static_cast<size_t>(0u);
  CUDA_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(_FD), &FD, sizeof(cuD*), off, hipMemcpyHostToDevice, s));
  CUDA_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(_FJ), &FJ, sizeof(cuJ*), off, hipMemcpyHostToDevice, s));
  CUDA_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(_GD), &GD, sizeof(cuD*), off, hipMemcpyHostToDevice, s));
  CUDA_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(_GJ), &GJ, sizeof(cuJ*), off, hipMemcpyHostToDevice, s));
  CUDA_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(_VD), &VD, sizeof(cuD*), off, hipMemcpyHostToDevice, s));
  CUDA_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(_VJ), &VJ, sizeof(cuJ*), off, hipMemcpyHostToDevice, s));
  CUDA_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(_S), &S, sizeof(double*), off, hipMemcpyHostToDevice, s));
  CUDA_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(_H), &H, sizeof(double*), off, hipMemcpyHostToDevice, s));
  CUDA_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(_K), &K, sizeof(double*), off, hipMemcpyHostToDevice, s));
  CUDA_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(_C), &C, sizeof(unsigned long long*), off, hipMemcpyHostToDevice, s));
  CUDA_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(_nRowF), &nRowF, sizeof(unsigned), off, hipMemcpyHostToDevice, s));
  CUDA_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(_nRowG), &nRowG, sizeof(unsigned), off, hipMemcpyHostToDevice, s));
  CUDA_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(_nRowV), &nRowV, sizeof(unsigned), off, hipMemcpyHostToDevice, s));
  CUDA_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(_nRank), &nRank, sizeof(unsigned), off, hipMemcpyHostToDevice, s));
  CUDA_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(_ldF), &ldF, sizeof(unsigned), off, hipMemcpyHostToDevice, s));
  CUDA_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(_ldG), &ldG, sizeof(unsigned), off, hipMemcpyHostToDevice, s));
  CUDA_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(_ldV), &ldV, sizeof(unsigned), off, hipMemcpyHostToDevice, s));
  CUDA_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(_nSwp), &nSwp, sizeof(unsigned), off, hipMemcpyHostToDevice, s));
  CUDA_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(_STRAT0_STEPS), &STRAT0_STEPS, sizeof(unsigned), off, hipMemcpyHostToDevice, s));
  CUDA_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(_STRAT0_PAIRS), &STRAT0_PAIRS, sizeof(unsigned), off, hipMemcpyHostToDevice, s));
  // copy strategy tables
  CUDA_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(_strat0), strat0, sizeof(strat0), off, hipMemcpyHostToDevice, s));
  CUDA_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(_strat1), strat1, sizeof(strat1), off, hipMemcpyHostToDevice, s));
}
