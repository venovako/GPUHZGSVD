#include "hip/hip_runtime.h"
#include "HZ.hpp"
#include "HZ_L.hpp"
#include "HZ_L2.hpp"

#include "device_code.hpp"
#include "cuda_memory_helper.hpp"

int // 0 if OK, < 0 if invalid argument, > 0 if error
HZ_L2_gpu
(const unsigned routine,    // IN, routine ID, <= 15, (B_N_)_2,
 // B: block-oriented (else, full-block), N: no sort;
 const unsigned ncol,       // IN, number of columns, <= min(nrowF, nrowG), == 0 (mod 32);
#ifdef ANIMATE
 const unsigned nrowF,      // IN, number of rows of F, == 0 (mod 64);
 const unsigned nrowG,      // IN, number of rows of G, == 0 (mod 64);
 cuD *const hFD,            // INOUT, ldhF x ncol host array in Fortran order;
 cuJ *const hFJ,            // INOUT, ldhF x ncol host array in Fortran order;
 const unsigned ldhF,       // IN, leading dimension of hF, >= nrowF;
 cuD *const dFD,            // INOUT, lddF x ncol device array in Fortran order;
 cuJ *const dFJ,            // INOUT, lddF x ncol device array in Fortran order;
 const unsigned lddF,       // IN, leading dimension of dF, >= nrowF;
 cuD *const hGD,            // INOUT, ldhG x ncol host array in Fortran order;
 cuJ *const hGJ,            // INOUT, ldhG x ncol host array in Fortran order;
 const unsigned ldhG,       // IN, leading dimension of hG, >= nrowG;
 cuD *const dGD,            // INOUT, lddG x ncol device array in Fortran order;
 cuJ *const dGJ,            // INOUT, lddG x ncol device array in Fortran order;
 const unsigned lddG,       // IN, leading dimension of dG, >= nrowG;
#endif // ANIMATE
 unsigned long long *const hC, // OUT, convergence vector
 unsigned long long *const dC, // OUT, convergence vector
 unsigned &glbSwp,          // OUT, number of sweeps at the outermost level;
 unsigned long long &glb_s, // OUT, number of rotations;
 unsigned long long &glb_b  // OUT, number of ``big'' rotations;
#ifdef ANIMATE
 , vn_cmplxvis_ctx *const ctx
 , std::complex<double> *const hDJ
 , const size_t nrow
#endif // ANIMATE
 ) throw()
{
  void (*const HZ_L1)(const unsigned) = ((routine & 2u) ? HZ_L1_v : HZ_L1_sv);

  const unsigned swp = ((routine & HZ_BO_2) ? 1u : HZ_NSWEEP);
  // stats per thread block
  const unsigned spb = 2u;
  // stats count
  const unsigned sc = STRAT1_PAIRS * spb;
  // stats len
  const size_t sl = sc * sizeof(unsigned long long);

  glb_s = 0ull;
  glb_b = 0ull;
#ifndef USE_MPI
  long long swp_tim = 0ll;
  stopwatch_reset(swp_tim);
#endif // !USE_MPI

  unsigned blk_swp = 0u;
  while (blk_swp < swp) {
    CUDA_CALL(hipMemset(dC, 0, sl));
    CUDA_CALL(hipDeviceSynchronize());
    for (unsigned blk_stp = 0u; blk_stp < STRAT1_STEPS; ++blk_stp) {
      if (blk_stp)
        CUDA_CALL(hipDeviceSynchronize());
      HZ_L1(blk_stp);
#ifdef ANIMATE
      if (ctx) {
        CUDA_CALL(hipDeviceSynchronize());

        CUDA_CALL(hipMemcpy2D(hFD, ldhF * sizeof(double), dFD, lddF * sizeof(cuD), nrowF * sizeof(cuD), ncol, hipMemcpyDeviceToHost));
        CUDA_CALL(hipMemcpy2D(hFJ, ldhF * sizeof(double), dFJ, lddF * sizeof(cuJ), nrowF * sizeof(cuJ), ncol, hipMemcpyDeviceToHost));
        CUDA_CALL(hipMemcpy2D(hGD, ldhG * sizeof(double), dGD, lddG * sizeof(cuD), nrowG * sizeof(cuD), ncol, hipMemcpyDeviceToHost));
        CUDA_CALL(hipMemcpy2D(hGJ, ldhG * sizeof(double), dGJ, lddG * sizeof(cuJ), nrowG * sizeof(cuJ), ncol, hipMemcpyDeviceToHost));
        CUDA_CALL(hipDeviceSynchronize());

        for (unsigned j = 0u; j < ncol; ++j) {
          const size_t offDJ = ldhDJ * j;
          const size_t offhF = ldhF * j;
          for (unsigned i = 0u; i < nrow; ++i) {
            const size_t ixDJ = offDJ + i;
            const size_t ixhF = offhF + i;
            hDJ[ixDJ].real(hFD[ixhF]);
            hDJ[ixDJ].imag(hFJ[ixhF]);
          }
        }
        SYSI_CALL(vn_cmplxvis_frame(ctx, (const vn_complex*)hDJ, nrow));

        for (unsigned j = 0u; j < ncol; ++j) {
          const size_t offDJ = ldhDJ * j;
          const size_t offhG = ldhG * j;
          for (unsigned i = 0u; i < nrow; ++i) {
            const size_t ixDJ = offDJ + i;
            const size_t ixhG = offhG + i;
            hDJ[ixDJ].real(hGD[ixhG]);
            hDJ[ixDJ].imag(hGJ[ixhG]);
          }
        }
        SYSI_CALL(vn_cmplxvis_frame(ctx, (const vn_complex*)hDJ, nrow));
      }
#endif // ANIMATE
    }

    CUDA_CALL(hipDeviceSynchronize());
    CUDA_CALL(hipMemcpy(hC, dC, sl, hipMemcpyDeviceToHost));
    CUDA_CALL(hipDeviceSynchronize());

    unsigned long long cvg_s = 0ull;
    unsigned long long cvg_b = 0ull;
    for (unsigned i = 0u; i < sc; i += spb) {
      cvg_s += hC[i];
      cvg_b += hC[i + 1u];
    }
    glb_s += cvg_s;
    glb_b += cvg_b;

#ifndef USE_MPI
    const double tim_s = stopwatch_lap(swp_tim) * TS2S;
    (void)fprintf(stdout, "BLK_SWP(%2u), ROT_S(%13llu), ROT_B(%13llu), TIME(%#14.6f s)\n", blk_swp, cvg_s, cvg_b, tim_s);
    (void)fflush(stdout);
#endif // !USE_MPI
    if (!cvg_b)
      break;
    ++blk_swp;
    initS(0, ncol);
    CUDA_CALL(hipDeviceSynchronize());
#ifdef ANIMATE
    if (ctx) {
      CUDA_CALL(hipMemcpy2D(hFD, ldhF * sizeof(double), dFD, lddF * sizeof(cuD), nrowF * sizeof(cuD), ncol, hipMemcpyDeviceToHost));
      CUDA_CALL(hipMemcpy2D(hFJ, ldhF * sizeof(double), dFJ, lddF * sizeof(cuJ), nrowF * sizeof(cuJ), ncol, hipMemcpyDeviceToHost));
      CUDA_CALL(hipMemcpy2D(hGD, ldhG * sizeof(double), dGD, lddG * sizeof(cuD), nrowG * sizeof(cuD), ncol, hipMemcpyDeviceToHost));
      CUDA_CALL(hipMemcpy2D(hGJ, ldhG * sizeof(double), dGJ, lddG * sizeof(cuJ), nrowG * sizeof(cuJ), ncol, hipMemcpyDeviceToHost));
      CUDA_CALL(hipDeviceSynchronize());

      for (unsigned j = 0u; j < ncol; ++j) {
        const size_t offDJ = ldhDJ * j;
        const size_t offhF = ldhF * j;
        for (unsigned i = 0u; i < nrow; ++i) {
          const size_t ixDJ = offDJ + i;
          const size_t ixhF = offhF + i;
          hDJ[ixDJ].real(hFD[ixhF]);
          hDJ[ixDJ].imag(hFJ[ixhF]);
        }
      }
      SYSI_CALL(vn_cmplxvis_frame(ctx, (const vn_complex*)hDJ, nrow));

      for (unsigned j = 0u; j < ncol; ++j) {
        const size_t offDJ = ldhDJ * j;
        const size_t offhG = ldhG * j;
        for (unsigned i = 0u; i < nrow; ++i) {
          const size_t ixDJ = offDJ + i;
          const size_t ixhG = offhG + i;
          hDJ[ixDJ].real(hGD[ixhG]);
          hDJ[ixDJ].imag(hGJ[ixhG]);
        }
      }
      SYSI_CALL(vn_cmplxvis_frame(ctx, (const vn_complex*)hDJ, nrow));
    }
#endif // ANIMATE
  }

  if (blk_swp < swp)
    glbSwp = (blk_swp + 1u);
  else
    glbSwp = blk_swp;
#ifdef USE_MPI
  if (blk_swp < swp)
    initS(0, ncol);
#else // !USE_MPI
  initS(1, ncol);
#endif // !USE_MPI
  CUDA_CALL(hipDeviceSynchronize());
  return 0;
}

int // 0 if OK, < 0 if invalid argument, > 0 if error
HZ_L2
(const unsigned routine,    // IN, routine ID, <= 15, (B_N_)_2,
 // B: block-oriented (else, full-block), N: no sort;
 const unsigned nrowF,      // IN, number of rows of F, == 0 (mod 64);
 const unsigned nrowG,      // IN, number of rows of G, == 0 (mod 64);
 const unsigned ncol,       // IN, number of columns, <= min(nrowF, nrowG), == 0 (mod 32);
 cuD *const hFD,            // INOUT, ldhF x ncol host array in Fortran order;
 cuJ *const hFJ,            // INOUT, ldhF x ncol host array in Fortran order;
 const unsigned ldhF,       // IN, leading dimension of F, >= nrowF;
 cuD *const hGD,            // INOUT, ldhG x ncol host array in Fortran order;
 cuJ *const hGJ,            // INOUT, ldhG x ncol host array in Fortran order;
 const unsigned ldhG,       // IN, leading dimension of G, >= nrowG;
 cuD *const hVD,            // INOUT, ldhV x ncol host array in Fortran order;
 cuJ *const hVJ,            // INOUT, ldhV x ncol host array in Fortran order;
 const unsigned ldhV,       // IN, leading dimension of V, >= ncol;
 double *const hS,          // OUT, the generalized singular values, optionally sorted in descending order;
 double *const hH,          // ||F_i||_F/sqrt(||F_i||_F^2 + ||G_i||_F^2);
 double *const hK,          // ||G_i||_F/sqrt(||F_i||_F^2 + ||G_i||_F^2);
 unsigned &glbSwp,          // OUT, number of sweeps at the outermost level;
 unsigned long long &glb_s, // OUT, number of rotations;
 unsigned long long &glb_b, // OUT, number of ``big'' rotations;
 double *const timing       // OUT, optional, in seconds, double[4] ==
 // WALL, SETUP & HOST ==> GPUs, COMPUTATION, CLEANUP & GPUs ==> HOST;
) throw()
{
  long long timers[4] = { 0ll };
  stopwatch_reset(timers[0]);

  if (routine >= 16u)
    return -1;

  if (!nrowF || (nrowF % 64u))
    return -2;
  if (!nrowG || (nrowG % 64u))
    return -3;
  if (!ncol || (ncol > nrowF) || (ncol > nrowG) || (ncol % 32u))
    return -4;

  if (!hFD)
    return -5;
  if (!hFJ)
    return -6;
  if (ldhF < nrowF)
    return -7;

  if (!hGD)
    return -8;
  if (!hGJ)
    return -9;
  if (ldhG < nrowG)
    return -10;

  if (!hVD)
    return -11;
  if (!hVJ)
    return -12;
  if (ldhV < ncol)
    return -13;

  if (!hS)
    return -14;
  if (!hH)
    return -15;
  if (!hK)
    return -16;

  stopwatch_reset(timers[3]);

  size_t lddF = static_cast<size_t>(nrowF);
  cuD *const dFD = allocDeviceMtx<cuD>(lddF, static_cast<size_t>(nrowF), static_cast<size_t>(ncol), true);
  cuJ *const dFJ = allocDeviceMtx<cuJ>(lddF, static_cast<size_t>(nrowF), static_cast<size_t>(ncol), true);

  size_t lddG = static_cast<size_t>(nrowG);
  cuD *const dGD = allocDeviceMtx<cuD>(lddG, static_cast<size_t>(nrowG), static_cast<size_t>(ncol), true);
  cuJ *const dGJ = allocDeviceMtx<cuJ>(lddG, static_cast<size_t>(nrowG), static_cast<size_t>(ncol), true);

  size_t lddV = static_cast<size_t>(ncol);
  cuD *const dVD = allocDeviceMtx<cuD>(lddV, static_cast<size_t>(ncol), static_cast<size_t>(ncol), true);
  cuJ *const dVJ = allocDeviceMtx<cuJ>(lddV, static_cast<size_t>(ncol), static_cast<size_t>(ncol), true);

  double *const dS = allocDeviceVec<double>(static_cast<size_t>(ncol));
  double *const dH = allocDeviceVec<double>(static_cast<size_t>(ncol));
  double *const dK = allocDeviceVec<double>(static_cast<size_t>(ncol));

  unsigned long long *const dC = allocDeviceVec<unsigned long long>(static_cast<size_t>(STRAT1_PAIRS) * 2u);
  unsigned long long *const hC = allocHostVec<unsigned long long>(static_cast<size_t>(STRAT1_PAIRS) * 2u);

  initSymbols(dFD,dFJ, dGD,dGJ, dVD,dVJ, dS,dH,dK, dC, nrowF,nrowG,ncol,ncol, lddF,lddG,lddV, ((routine & HZ_BO_1) ? 1u : HZ_NSWEEP));
  CUDA_CALL(hipMemcpy2D(dFD, lddF * sizeof(cuD), hFD, ldhF * sizeof(double), nrowF * sizeof(cuD), ncol, hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy2D(dFJ, lddF * sizeof(cuJ), hFJ, ldhF * sizeof(double), nrowF * sizeof(cuJ), ncol, hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy2D(dGD, lddG * sizeof(cuD), hGD, ldhG * sizeof(double), nrowG * sizeof(cuD), ncol, hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy2D(dGJ, lddG * sizeof(cuJ), hGJ, ldhG * sizeof(double), nrowG * sizeof(cuJ), ncol, hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy2D(dVD, lddV * sizeof(cuD), hVD, ldhV * sizeof(double), ncol * sizeof(cuD), ncol, hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy2D(dVJ, lddV * sizeof(cuJ), hVJ, ldhV * sizeof(double), ncol * sizeof(cuJ), ncol, hipMemcpyHostToDevice));
  CUDA_CALL(hipDeviceSynchronize());
#ifdef USE_MPI
  const unsigned ifc0 = 0u;
  const unsigned ifc1 = (ncol >> 1u);
  initV(((CVG == 0) || (CVG == 1) || (CVG == 4) || (CVG == 5)), ncol, ifc0, ifc1);
#else // !USE_MPI
  initV(((CVG == 0) || (CVG == 1) || (CVG == 4) || (CVG == 5)), ncol);
#endif // USE_MPI
  CUDA_CALL(hipDeviceSynchronize());

#ifdef ANIMATE
  vn_cmplxvis_ctx *ctx = static_cast<vn_cmplxvis_ctx*>(NULL);
  std::complex<double> *hDJ = static_cast<std::complex<double>>(NULL);
  size_t nrow = 0u;
  // it is meant to work only for nrowF == nrowG
  if (nrowF == nrowG) {
    nrow = nrowF;
    hDJ = allocHostMtx<std::complex<double>>(nrow, nrow, static_cast<size_t>(ncol), true);
  }
  if (ncol < 10000u) {
    char fname[8] = { '\0' };
    (void)sprintf(fname, "FG%x%04u", routine, ncol);
    if (hDJ)
      SYSI_CALL(vn_cmplxvis_start(&ctx, fname, (VN_CMPLXVIS_OP_AhA | VN_CMPLXVIS_FN_Lg), ncol, ncol, 1, 1, 7));
    if (ctx) {
      CUDA_CALL(hipMemcpy2D(hFD, ldhF * sizeof(double), dFD, lddF * sizeof(cuD), nrowF * sizeof(cuD), ncol, hipMemcpyDeviceToHost));
      CUDA_CALL(hipMemcpy2D(hFJ, ldhF * sizeof(double), dFJ, lddF * sizeof(cuJ), nrowF * sizeof(cuJ), ncol, hipMemcpyDeviceToHost));
      CUDA_CALL(hipMemcpy2D(hGD, ldhG * sizeof(double), dGD, lddG * sizeof(cuD), nrowG * sizeof(cuD), ncol, hipMemcpyDeviceToHost));
      CUDA_CALL(hipMemcpy2D(hGJ, ldhG * sizeof(double), dGJ, lddG * sizeof(cuJ), nrowG * sizeof(cuJ), ncol, hipMemcpyDeviceToHost));
      CUDA_CALL(hipDeviceSynchronize());

      for (unsigned j = 0u; j < ncol; ++j) {
        const size_t offDJ = ldhDJ * j;
        const size_t offhF = ldhF * j;
        for (unsigned i = 0u; i < nrow; ++i) {
          const size_t ixDJ = offDJ + i;
          const size_t ixhF = offhF + i;
          hDJ[ixDJ].real(hFD[ixhF]);
          hDJ[ixDJ].imag(hFJ[ixhF]);
        }
      }
      SYSI_CALL(vn_cmplxvis_frame(ctx, (const vn_complex*)hDJ, nrow));

      for (unsigned j = 0u; j < ncol; ++j) {
        const size_t offDJ = ldhDJ * j;
        const size_t offhG = ldhG * j;
        for (unsigned i = 0u; i < nrow; ++i) {
          const size_t ixDJ = offDJ + i;
          const size_t ixhG = offhG + i;
          hDJ[ixDJ].real(hGD[ixhG]);
          hDJ[ixDJ].imag(hGJ[ixhG]);
        }
      }
      SYSI_CALL(vn_cmplxvis_frame(ctx, (const vn_complex*)hDJ, nrow));
    }
  }
#endif // ANIMATE

  timers[1] = stopwatch_lap(timers[3]);
  const int ret = HZ_L2_gpu
    (routine,ncol,
#ifdef ANIMATE
     nrowF,nrowG, hFD,hFJ,ldhF, dFD,dFJ,lddF, hGD,hGJ,ldhG, dGD,dGJ,lddG,
#endif // ANIMATE
     hC,dC, glbSwp,glb_s,glb_b
#ifdef ANIMATE
     , ctx,hDJ,nrow
#endif // ANIMATE
     );
  timers[2] = stopwatch_lap(timers[3]);

  CUDA_CALL(hipMemcpy2D(hFD, ldhF * sizeof(double), dFD, lddF * sizeof(cuD), nrowF * sizeof(cuD), ncol, hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy2D(hFJ, ldhF * sizeof(double), dFJ, lddF * sizeof(cuJ), nrowF * sizeof(cuJ), ncol, hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy2D(hGD, ldhG * sizeof(double), dGD, lddG * sizeof(cuD), nrowG * sizeof(cuD), ncol, hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy2D(hGJ, ldhG * sizeof(double), dGJ, lddG * sizeof(cuJ), nrowG * sizeof(cuJ), ncol, hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy2D(hVD, ldhV * sizeof(double), dVD, lddV * sizeof(cuD), ncol * sizeof(cuD), ncol, hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy2D(hVJ, ldhV * sizeof(double), dVJ, lddV * sizeof(cuJ), ncol * sizeof(cuJ), ncol, hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(hS, dS, ncol * sizeof(double), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(hH, dH, ncol * sizeof(double), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(hK, dK, ncol * sizeof(double), hipMemcpyDeviceToHost));
  CUDA_CALL(hipDeviceSynchronize());

#ifdef ANIMATE
  if (ctx) {
    for (unsigned j = 0u; j < ncol; ++j) {
      const size_t offDJ = ldhDJ * j;
      const size_t offhF = ldhF * j;
      for (unsigned i = 0u; i < nrow; ++i) {
        const size_t ixDJ = offDJ + i;
        const size_t ixhF = offhF + i;
        hDJ[ixDJ].real(hFD[ixhF]);
        hDJ[ixDJ].imag(hFJ[ixhF]);
      }
    }
    SYSI_CALL(vn_cmplxvis_frame(ctx, (const vn_complex*)hDJ, nrow));

    for (unsigned j = 0u; j < ncol; ++j) {
      const size_t offDJ = ldhDJ * j;
      const size_t offhG = ldhG * j;
      for (unsigned i = 0u; i < nrow; ++i) {
        const size_t ixDJ = offDJ + i;
        const size_t ixhG = offhG + i;
        hDJ[ixDJ].real(hGD[ixhG]);
        hDJ[ixDJ].imag(hGJ[ixhG]);
      }
    }
    SYSI_CALL(vn_cmplxvis_frame(ctx, (const vn_complex*)hDJ, nrow));

    SYSI_CALL(vn_cmplxvis_stop(ctx));
    CUDA_CALL(hipHostFree((void*)hDJ));
  }
#endif // ANIMATE

  CUDA_CALL(hipHostFree(hC));
  CUDA_CALL(hipFree(dC));
  CUDA_CALL(hipFree(dK));
  CUDA_CALL(hipFree(dH));
  CUDA_CALL(hipFree(dS));
  CUDA_CALL(hipFree(dVJ));
  CUDA_CALL(hipFree(dVD));
  CUDA_CALL(hipFree(dGJ));
  CUDA_CALL(hipFree(dGD));
  CUDA_CALL(hipFree(dFJ));
  CUDA_CALL(hipFree(dFD));
  CUDA_CALL(hipDeviceSynchronize());

  timers[3] = stopwatch_lap(timers[3]);
  timers[0] = stopwatch_lap(timers[0]);

  if (timing)
    for (unsigned i = 0u; i < 4u; ++i)
      timing[i] = timers[i] * TS2S;

  return ret;
}
