#include "hip/hip_runtime.h"
#include "HZ.hpp"
#include "HZ_L.hpp"
#include "HZ_L2.hpp"

#include "device_code.hpp"
#include "cuda_memory_helper.hpp"

int // 0 if OK, < 0 if invalid argument, > 0 if error
HZ_L2_gpu
(const unsigned routine,    // IN, routine ID, <= 15, (B_N_)_2,
 // B: block-oriented (else, full-block), N: no sort;
 const unsigned ncol,       // IN, number of columns, <= min(nrowF, nrowG), == 0 (mod 32);
#ifdef ANIMATE
 const unsigned nrowF,      // IN, number of rows of F, == 0 (mod 64);
 const unsigned nrowG,      // IN, number of rows of G, == 0 (mod 64);
 cuD *const hFD,            // INOUT, ldhF x ncol host array in Fortran order;
 cuJ *const hFJ,            // INOUT, ldhF x ncol host array in Fortran order;
 const unsigned ldhF,       // IN, leading dimension of hF, >= nrowF;
 cuD *const dFD,            // INOUT, lddF x ncol device array in Fortran order;
 cuJ *const dFJ,            // INOUT, lddF x ncol device array in Fortran order;
 const unsigned lddF,       // IN, leading dimension of dF, >= nrowF;
 cuD *const hGD,            // INOUT, ldhG x ncol host array in Fortran order;
 cuJ *const hGJ,            // INOUT, ldhG x ncol host array in Fortran order;
 const unsigned ldhG,       // IN, leading dimension of hG, >= nrowG;
 cuD *const dGD,            // INOUT, lddG x ncol device array in Fortran order;
 cuJ *const dGJ,            // INOUT, lddG x ncol device array in Fortran order;
 const unsigned lddG,       // IN, leading dimension of dG, >= nrowG;
#endif /* ANIMATE */
 unsigned long long *const hC, // OUT, convergence vector
 unsigned long long *const dC, // OUT, convergence vector
 unsigned &glbSwp,          // OUT, number of sweeps at the outermost level;
 unsigned long long &glb_s, // OUT, number of rotations;
 unsigned long long &glb_b  // OUT, number of ``big'' rotations;
#ifdef ANIMATE
 , vn_cmplxvis_ctx *const ctx
 , std::complex<double> *const hDJ
 , const size_t nrow
#endif /* ANIMATE */
 , const hipStream_t s
) throw()
{
  void (*const HZ_L1)(const unsigned, const hipStream_t) = ((routine & 2u) ? HZ_L1_v : HZ_L1_sv);

  const unsigned swp = ((routine & HZ_BO_2) ? 1u : HZ_NSWEEP);
  // stats count
  const unsigned sc = STRAT1_PAIRS * C_ELEMS_PER_BLOCK;
  // stats len
  const size_t sl = sc * sizeof(unsigned long long);

  glb_s = 0ull;
  glb_b = 0ull;
#if (defined(PROFILE) && (PROFILE == 0))
  unsigned long long CLK_1 = 0ull;
  unsigned long long CLK_2 = 0ull;
  unsigned long long CLK_3 = 0ull;
  unsigned long long CLK_4 = 0ull;
#endif /* ?PROFILE */

#ifndef USE_MPI
  long long swp_tim = 0ll;
  stopwatch_reset(swp_tim);
#endif /* !USE_MPI */

  unsigned blk_swp = 0u;
  while (blk_swp < swp) {
    CUDA_CALL(hipMemsetAsync(dC, 0, sl, s));
    CUDA_CALL(hipStreamSynchronize(s));
    for (unsigned blk_stp = 0u; blk_stp < STRAT1_STEPS; ++blk_stp) {
      if (blk_stp)
        CUDA_CALL(hipStreamSynchronize(s));
      HZ_L1(blk_stp, s);

#ifdef ANIMATE
      if (ctx) {
        CUDA_CALL(hipStreamSynchronize(s));

        CUDA_CALL(hipMemcpy2DAsync(hFD, ldhF * sizeof(double), dFD, lddF * sizeof(cuD), nrowF * sizeof(cuD), ncol, hipMemcpyDeviceToHost, s));
        CUDA_CALL(hipMemcpy2DAsync(hFJ, ldhF * sizeof(double), dFJ, lddF * sizeof(cuJ), nrowF * sizeof(cuJ), ncol, hipMemcpyDeviceToHost, s));
        CUDA_CALL(hipMemcpy2DAsync(hGD, ldhG * sizeof(double), dGD, lddG * sizeof(cuD), nrowG * sizeof(cuD), ncol, hipMemcpyDeviceToHost, s));
        CUDA_CALL(hipMemcpy2DAsync(hGJ, ldhG * sizeof(double), dGJ, lddG * sizeof(cuJ), nrowG * sizeof(cuJ), ncol, hipMemcpyDeviceToHost, s));
        CUDA_CALL(hipStreamSynchronize(s));

        for (unsigned j = 0u; j < ncol; ++j) {
          const size_t offDJ = ldhDJ * j;
          const size_t offhF = ldhF * j;
          for (unsigned i = 0u; i < nrow; ++i) {
            const size_t ixDJ = offDJ + i;
            const size_t ixhF = offhF + i;
            hDJ[ixDJ].real(hFD[ixhF]);
            hDJ[ixDJ].imag(hFJ[ixhF]);
          }
        }
        SYSI_CALL(vn_cmplxvis_frame(ctx, (const vn_complex*)hDJ, nrow));

        for (unsigned j = 0u; j < ncol; ++j) {
          const size_t offDJ = ldhDJ * j;
          const size_t offhG = ldhG * j;
          for (unsigned i = 0u; i < nrow; ++i) {
            const size_t ixDJ = offDJ + i;
            const size_t ixhG = offhG + i;
            hDJ[ixDJ].real(hGD[ixhG]);
            hDJ[ixDJ].imag(hGJ[ixhG]);
          }
        }
        SYSI_CALL(vn_cmplxvis_frame(ctx, (const vn_complex*)hDJ, nrow));
      }
#endif /* ANIMATE */
    }

    CUDA_CALL(hipStreamSynchronize(s));
    CUDA_CALL(hipMemcpyAsync(hC, dC, sl, hipMemcpyDeviceToHost, s));
    CUDA_CALL(hipStreamSynchronize(s));

    unsigned long long cvg_s = 0ull;
    unsigned long long cvg_b = 0ull;
#if (defined(PROFILE) && (PROFILE == 0))
    unsigned long long clk_1 = 0ull;
    unsigned long long clk_2 = 0ull;
    unsigned long long clk_3 = 0ull;
    unsigned long long clk_4 = 0ull;
#endif /* ?PROFILE */
    for (unsigned i = 0u; i < sc; i += C_ELEMS_PER_BLOCK) {
      cvg_s += hC[i + C_SMALL];
      cvg_b += hC[i + C_BIG];
#if (defined(PROFILE) && (PROFILE == 0))
      if (clk_1 < hC[i + C_SUBPHASE_1])
        clk_1 = hC[i + C_SUBPHASE_1];
      if (clk_2 < hC[i + C_SUBPHASE_2])
        clk_2 = hC[i + C_SUBPHASE_2];
      if (clk_3 < hC[i + C_SUBPHASE_3])
        clk_3 = hC[i + C_SUBPHASE_3];
      if (clk_4 < hC[i + C_SUBPHASE_4])
        clk_4 = hC[i + C_SUBPHASE_4];
#endif /* ?PROFILE */
    }
    glb_s += cvg_s;
    glb_b += cvg_b;
#if (defined(PROFILE) && (PROFILE == 0))
    CLK_1 += clk_1;
    CLK_2 += clk_2;
    CLK_3 += clk_3;
    CLK_4 += clk_4;
#endif /* ?PROFILE */

#ifndef USE_MPI
    const double tim_s = stopwatch_lap(swp_tim) * TS2S;
    (void)fprintf(stdout, "BLK_SWP(%2u), ROT_S(%13llu), ROT_B(%13llu), TIME(%#14.6f s)", blk_swp, cvg_s, cvg_b, tim_s);
#if (defined(PROFILE) && (PROFILE == 0))
    (void)fprintf(stdout, ", clk_1(%11llu), clk_2(%11llu), clk_3(%11llu), clk_4(%11llu)", clk_1, clk_2, clk_3, clk_4);
#endif /* ?PROFILE */
    (void)fprintf(stdout, "\n");
    (void)fflush(stdout);
#endif /* !USE_MPI */
    if (!cvg_b)
      break;
    ++blk_swp;
    initS(0, ncol, s);
    CUDA_CALL(hipStreamSynchronize(s));

#ifdef ANIMATE
    if (ctx) {
      CUDA_CALL(hipMemcpy2DAsync(hFD, ldhF * sizeof(double), dFD, lddF * sizeof(cuD), nrowF * sizeof(cuD), ncol, hipMemcpyDeviceToHost, s));
      CUDA_CALL(hipMemcpy2DAsync(hFJ, ldhF * sizeof(double), dFJ, lddF * sizeof(cuJ), nrowF * sizeof(cuJ), ncol, hipMemcpyDeviceToHost, s));
      CUDA_CALL(hipMemcpy2DAsync(hGD, ldhG * sizeof(double), dGD, lddG * sizeof(cuD), nrowG * sizeof(cuD), ncol, hipMemcpyDeviceToHost, s));
      CUDA_CALL(hipMemcpy2DAsync(hGJ, ldhG * sizeof(double), dGJ, lddG * sizeof(cuJ), nrowG * sizeof(cuJ), ncol, hipMemcpyDeviceToHost, s));
      CUDA_CALL(hipStreamSynchronize(s));

      for (unsigned j = 0u; j < ncol; ++j) {
        const size_t offDJ = ldhDJ * j;
        const size_t offhF = ldhF * j;
        for (unsigned i = 0u; i < nrow; ++i) {
          const size_t ixDJ = offDJ + i;
          const size_t ixhF = offhF + i;
          hDJ[ixDJ].real(hFD[ixhF]);
          hDJ[ixDJ].imag(hFJ[ixhF]);
        }
      }
      SYSI_CALL(vn_cmplxvis_frame(ctx, (const vn_complex*)hDJ, nrow));

      for (unsigned j = 0u; j < ncol; ++j) {
        const size_t offDJ = ldhDJ * j;
        const size_t offhG = ldhG * j;
        for (unsigned i = 0u; i < nrow; ++i) {
          const size_t ixDJ = offDJ + i;
          const size_t ixhG = offhG + i;
          hDJ[ixDJ].real(hGD[ixhG]);
          hDJ[ixDJ].imag(hGJ[ixhG]);
        }
      }
      SYSI_CALL(vn_cmplxvis_frame(ctx, (const vn_complex*)hDJ, nrow));
    }
#endif /* ANIMATE */
  }

  if (blk_swp < swp)
    glbSwp = (blk_swp + 1u);
  else
    glbSwp = blk_swp;
#ifdef USE_MPI
  if (blk_swp < swp)
    initS(0, ncol, s);
#else /* !USE_MPI */
  initS(1, ncol, s);
#endif /* ?USE_MPI */
  CUDA_CALL(hipStreamSynchronize(s));

#if (defined(PROFILE) && (PROFILE == 0))
  (void)fprintf(stdout, "CLK_1(%13llu), CLK_2(%13llu), CLK_3(%13llu), CLK_4(%13llu)\n", CLK_1, CLK_2, CLK_3, CLK_4);
  (void)fflush(stdout);
#endif /* ?PROFILE */

  return 0;
}

int // 0 if OK, < 0 if invalid argument, > 0 if error
HZ_L2
(const unsigned routine,    // IN, routine ID, <= 15, (B_N_)_2,
 // B: block-oriented (else, full-block), N: no sort;
 const unsigned nrowF,      // IN, number of rows of F, == 0 (mod 64);
 const unsigned nrowG,      // IN, number of rows of G, == 0 (mod 64);
 const unsigned ncol,       // IN, number of columns, <= min(nrowF, nrowG), == 0 (mod 32);
 cuD *const hFD,            // INOUT, ldhF x ncol host array in Fortran order;
 cuJ *const hFJ,            // INOUT, ldhF x ncol host array in Fortran order;
 const unsigned ldhF,       // IN, leading dimension of F, >= nrowF;
 cuD *const hGD,            // INOUT, ldhG x ncol host array in Fortran order;
 cuJ *const hGJ,            // INOUT, ldhG x ncol host array in Fortran order;
 const unsigned ldhG,       // IN, leading dimension of G, >= nrowG;
 cuD *const hVD,            // INOUT, ldhV x ncol host array in Fortran order;
 cuJ *const hVJ,            // INOUT, ldhV x ncol host array in Fortran order;
 const unsigned ldhV,       // IN, leading dimension of V, >= ncol;
 double *const hS,          // OUT, the generalized singular values, optionally sorted in descending order;
 double *const hH,          // ||F_i||_F/sqrt(||F_i||_F^2 + ||G_i||_F^2);
 double *const hK,          // ||G_i||_F/sqrt(||F_i||_F^2 + ||G_i||_F^2);
 unsigned &glbSwp,          // OUT, number of sweeps at the outermost level;
 unsigned long long &glb_s, // OUT, number of rotations;
 unsigned long long &glb_b, // OUT, number of ``big'' rotations;
 double *const timing,      // OUT, optional, in seconds, double[4] ==
 // WALL, SETUP & HOST ==> GPUs, COMPUTATION, CLEANUP & GPUs ==> HOST;
 const hipStream_t s
) throw()
{
  long long timers[4] = { 0ll };
  stopwatch_reset(timers[0]);

  if (routine >= 16u)
    return -1;

  if (!nrowF || (nrowF % 64u))
    return -2;
  if (!nrowG || (nrowG % 64u))
    return -3;
  if (!ncol || (ncol > nrowF) || (ncol > nrowG) || (ncol % 32u))
    return -4;

  if (!hFD)
    return -5;
  if (!hFJ)
    return -6;
  if (ldhF < nrowF)
    return -7;

  if (!hGD)
    return -8;
  if (!hGJ)
    return -9;
  if (ldhG < nrowG)
    return -10;

  if (!hVD)
    return -11;
  if (!hVJ)
    return -12;
  if (ldhV < ncol)
    return -13;

  if (!hS)
    return -14;
  if (!hH)
    return -15;
  if (!hK)
    return -16;

  stopwatch_reset(timers[3]);

  size_t lddF = static_cast<size_t>(nrowF);
  cuD *const dFD = allocDeviceMtx<cuD>(lddF, static_cast<size_t>(nrowF), static_cast<size_t>(ncol), true, s);
  cuJ *const dFJ = allocDeviceMtx<cuJ>(lddF, static_cast<size_t>(nrowF), static_cast<size_t>(ncol), true, s);

  size_t lddG = static_cast<size_t>(nrowG);
  cuD *const dGD = allocDeviceMtx<cuD>(lddG, static_cast<size_t>(nrowG), static_cast<size_t>(ncol), true, s);
  cuJ *const dGJ = allocDeviceMtx<cuJ>(lddG, static_cast<size_t>(nrowG), static_cast<size_t>(ncol), true, s);

  size_t lddV = static_cast<size_t>(ncol);
  cuD *const dVD = allocDeviceMtx<cuD>(lddV, static_cast<size_t>(ncol), static_cast<size_t>(ncol), true, s);
  cuJ *const dVJ = allocDeviceMtx<cuJ>(lddV, static_cast<size_t>(ncol), static_cast<size_t>(ncol), true, s);

  double *const dS = allocDeviceVec<double>(static_cast<size_t>(ncol), s);
  double *const dH = allocDeviceVec<double>(static_cast<size_t>(ncol), s);
  double *const dK = allocDeviceVec<double>(static_cast<size_t>(ncol), s);

  unsigned long long *const dC = allocDeviceVec<unsigned long long>((static_cast<size_t>(STRAT1_PAIRS) * C_ELEMS_PER_BLOCK), s);
  unsigned long long *const hC = allocHostVec<unsigned long long>(static_cast<size_t>(STRAT1_PAIRS) * C_ELEMS_PER_BLOCK);

  initSymbols(dFD,dFJ, dGD,dGJ, dVD,dVJ, dS,dH,dK, dC, nrowF,nrowG,ncol,ncol, lddF,lddG,lddV, ((routine & HZ_BO_1) ? 1u : HZ_NSWEEP), s);
  CUDA_CALL(hipMemcpy2DAsync(dFD, lddF * sizeof(cuD), hFD, ldhF * sizeof(double), nrowF * sizeof(cuD), ncol, hipMemcpyHostToDevice, s));
  CUDA_CALL(hipMemcpy2DAsync(dFJ, lddF * sizeof(cuJ), hFJ, ldhF * sizeof(double), nrowF * sizeof(cuJ), ncol, hipMemcpyHostToDevice, s));
  CUDA_CALL(hipMemcpy2DAsync(dGD, lddG * sizeof(cuD), hGD, ldhG * sizeof(double), nrowG * sizeof(cuD), ncol, hipMemcpyHostToDevice, s));
  CUDA_CALL(hipMemcpy2DAsync(dGJ, lddG * sizeof(cuJ), hGJ, ldhG * sizeof(double), nrowG * sizeof(cuJ), ncol, hipMemcpyHostToDevice, s));
  CUDA_CALL(hipMemcpy2DAsync(dVD, lddV * sizeof(cuD), hVD, ldhV * sizeof(double), ncol * sizeof(cuD), ncol, hipMemcpyHostToDevice, s));
  CUDA_CALL(hipMemcpy2DAsync(dVJ, lddV * sizeof(cuJ), hVJ, ldhV * sizeof(double), ncol * sizeof(cuJ), ncol, hipMemcpyHostToDevice, s));
  CUDA_CALL(hipStreamSynchronize(s));
#ifndef USE_MPI
  cuda_prof_start();
#endif /* !USE_MPI */

#ifdef USE_MPI
  const unsigned ifc0 = 0u;
  const unsigned ifc1 = (ncol >> 1u);
  initV(((CVG == 0) || (CVG == 1) || (CVG == 4) || (CVG == 5)), ncol, ifc0, ifc1, s);
#else /* !USE_MPI */
  initV(((CVG == 0) || (CVG == 1) || (CVG == 4) || (CVG == 5)), ncol, s);
#endif /* ?USE_MPI */
  CUDA_CALL(hipStreamSynchronize(s));

#ifdef ANIMATE
  vn_cmplxvis_ctx *ctx = static_cast<vn_cmplxvis_ctx*>(NULL);
  std::complex<double> *hDJ = static_cast<std::complex<double>>(NULL);
  size_t nrow = 0u;
  // it is meant to work only for nrowF == nrowG
  if (nrowF == nrowG) {
    nrow = nrowF;
    hDJ = allocHostMtx<std::complex<double>>(nrow, nrow, static_cast<size_t>(ncol), true);
  }
  if (ncol < 10000u) {
    char fname[8] = { '\0' };
    (void)sprintf(fname, "FG%x%04u", routine, ncol);
    if (hDJ)
      SYSI_CALL(vn_cmplxvis_start(&ctx, fname, (VN_CMPLXVIS_OP_AhA | VN_CMPLXVIS_FN_Lg), ncol, ncol, 1, 1, 7));
    if (ctx) {
      CUDA_CALL(hipMemcpy2DAsync(hFD, ldhF * sizeof(double), dFD, lddF * sizeof(cuD), nrowF * sizeof(cuD), ncol, hipMemcpyDeviceToHost, s));
      CUDA_CALL(hipMemcpy2DAsync(hFJ, ldhF * sizeof(double), dFJ, lddF * sizeof(cuJ), nrowF * sizeof(cuJ), ncol, hipMemcpyDeviceToHost, s));
      CUDA_CALL(hipMemcpy2DAsync(hGD, ldhG * sizeof(double), dGD, lddG * sizeof(cuD), nrowG * sizeof(cuD), ncol, hipMemcpyDeviceToHost, s));
      CUDA_CALL(hipMemcpy2DAsync(hGJ, ldhG * sizeof(double), dGJ, lddG * sizeof(cuJ), nrowG * sizeof(cuJ), ncol, hipMemcpyDeviceToHost, s));
      CUDA_CALL(hipStreamSynchronize(s));

      for (unsigned j = 0u; j < ncol; ++j) {
        const size_t offDJ = ldhDJ * j;
        const size_t offhF = ldhF * j;
        for (unsigned i = 0u; i < nrow; ++i) {
          const size_t ixDJ = offDJ + i;
          const size_t ixhF = offhF + i;
          hDJ[ixDJ].real(hFD[ixhF]);
          hDJ[ixDJ].imag(hFJ[ixhF]);
        }
      }
      SYSI_CALL(vn_cmplxvis_frame(ctx, (const vn_complex*)hDJ, nrow));

      for (unsigned j = 0u; j < ncol; ++j) {
        const size_t offDJ = ldhDJ * j;
        const size_t offhG = ldhG * j;
        for (unsigned i = 0u; i < nrow; ++i) {
          const size_t ixDJ = offDJ + i;
          const size_t ixhG = offhG + i;
          hDJ[ixDJ].real(hGD[ixhG]);
          hDJ[ixDJ].imag(hGJ[ixhG]);
        }
      }
      SYSI_CALL(vn_cmplxvis_frame(ctx, (const vn_complex*)hDJ, nrow));
    }
  }
#endif /* ANIMATE */

  timers[1] = stopwatch_lap(timers[3]);
  const int ret = HZ_L2_gpu
    (routine,ncol
#ifdef ANIMATE
     , nrowF,nrowG, hFD,hFJ,ldhF, dFD,dFJ,lddF, hGD,hGJ,ldhG, dGD,dGJ,lddG,
#endif /* ANIMATE */
     , hC,dC, glbSwp,glb_s,glb_b
#ifdef ANIMATE
     , ctx,hDJ,nrow
#endif /* ANIMATE */
     , s
     );
  timers[2] = stopwatch_lap(timers[3]);
#ifndef USE_MPI
  cuda_prof_stop();
#endif /* !USE_MPI */

  CUDA_CALL(hipMemcpy2DAsync(hFD, ldhF * sizeof(double), dFD, lddF * sizeof(cuD), nrowF * sizeof(cuD), ncol, hipMemcpyDeviceToHost, s));
  CUDA_CALL(hipMemcpy2DAsync(hFJ, ldhF * sizeof(double), dFJ, lddF * sizeof(cuJ), nrowF * sizeof(cuJ), ncol, hipMemcpyDeviceToHost, s));
  CUDA_CALL(hipMemcpy2DAsync(hGD, ldhG * sizeof(double), dGD, lddG * sizeof(cuD), nrowG * sizeof(cuD), ncol, hipMemcpyDeviceToHost, s));
  CUDA_CALL(hipMemcpy2DAsync(hGJ, ldhG * sizeof(double), dGJ, lddG * sizeof(cuJ), nrowG * sizeof(cuJ), ncol, hipMemcpyDeviceToHost, s));
  CUDA_CALL(hipMemcpy2DAsync(hVD, ldhV * sizeof(double), dVD, lddV * sizeof(cuD), ncol * sizeof(cuD), ncol, hipMemcpyDeviceToHost, s));
  CUDA_CALL(hipMemcpy2DAsync(hVJ, ldhV * sizeof(double), dVJ, lddV * sizeof(cuJ), ncol * sizeof(cuJ), ncol, hipMemcpyDeviceToHost, s));
  CUDA_CALL(hipMemcpyAsync(hS, dS, ncol * sizeof(double), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpyAsync(hH, dH, ncol * sizeof(double), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpyAsync(hK, dK, ncol * sizeof(double), hipMemcpyDeviceToHost));
  CUDA_CALL(hipStreamSynchronize(s));

#ifdef ANIMATE
  if (ctx) {
    for (unsigned j = 0u; j < ncol; ++j) {
      const size_t offDJ = ldhDJ * j;
      const size_t offhF = ldhF * j;
      for (unsigned i = 0u; i < nrow; ++i) {
        const size_t ixDJ = offDJ + i;
        const size_t ixhF = offhF + i;
        hDJ[ixDJ].real(hFD[ixhF]);
        hDJ[ixDJ].imag(hFJ[ixhF]);
      }
    }
    SYSI_CALL(vn_cmplxvis_frame(ctx, (const vn_complex*)hDJ, nrow));

    for (unsigned j = 0u; j < ncol; ++j) {
      const size_t offDJ = ldhDJ * j;
      const size_t offhG = ldhG * j;
      for (unsigned i = 0u; i < nrow; ++i) {
        const size_t ixDJ = offDJ + i;
        const size_t ixhG = offhG + i;
        hDJ[ixDJ].real(hGD[ixhG]);
        hDJ[ixDJ].imag(hGJ[ixhG]);
      }
    }
    SYSI_CALL(vn_cmplxvis_frame(ctx, (const vn_complex*)hDJ, nrow));

    SYSI_CALL(vn_cmplxvis_stop(ctx));
    CUDA_CALL(hipHostFree((void*)hDJ));
  }
#endif /* ANIMATE */

  CUDA_CALL(hipHostFree(hC));
  CUDA_CALL(hipFree(dC));
  CUDA_CALL(hipFree(dK));
  CUDA_CALL(hipFree(dH));
  CUDA_CALL(hipFree(dS));
  CUDA_CALL(hipFree(dVJ));
  CUDA_CALL(hipFree(dVD));
  CUDA_CALL(hipFree(dGJ));
  CUDA_CALL(hipFree(dGD));
  CUDA_CALL(hipFree(dFJ));
  CUDA_CALL(hipFree(dFD));

  timers[3] = stopwatch_lap(timers[3]);
  timers[0] = stopwatch_lap(timers[0]);

  if (timing)
    for (unsigned i = 0u; i < 4u; ++i)
      timing[i] = timers[i] * TS2S;

  return ret;
}
