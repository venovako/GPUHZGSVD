#include "HZ.hpp"

#include "device_code.hpp"
#include "device_code_common.hpp"
#include "device_code_accumV.hpp"
#if (CVG == 0)
#include "device_code_cdsort_0.hpp"
#elif (CVG == 1)
#include "device_code_cdsort_1.hpp"
#elif (CVG == 2)
#include "device_code_cdsort_2.hpp"
#elif (CVG == 3)
#include "device_code_cdsort_3.hpp"
#elif (CVG == 4)
#include "device_code_cdsort_4.hpp"
#elif (CVG == 5)
#include "device_code_cdsort_5.hpp"
#elif (CVG == 6)
#include "device_code_cdsort_6.hpp"
#elif (CVG == 7)
#include "device_code_cdsort_7.hpp"
#else // unknown CVG
#error CVG unknown
#endif // ?CVG
#include "device_code_cdsort_accumV.hpp"

#include "my_utils.hpp"

static const dim3 hzL1bD(HZ_L1_THREADS_PER_BLOCK_X, HZ_L1_THREADS_PER_BLOCK_Y, 1u);

void HZ_L1_sv(const unsigned step) throw()
{
  const dim3 hzL1gD(STRAT1_PAIRS, 1u, 1u);
  dHZ_L1_sv<<< hzL1gD, hzL1bD >>>(step);
}

void initS(const int full, const unsigned nRank) throw()
{
  const dim3 bD(2u * WARP_SZ, 1u, 1u);
  const dim3 gD(udiv_ceil(nRank * WARP_SZ, bD.x), 1u, 1u);
  const size_t shmD = static_cast<size_t>(0u);
  dInitS<<< gD, bD, shmD >>>(full);
}

void initV(const int sclV, const unsigned nRank) throw()
{
  const dim3 bD(2u * WARP_SZ, 1u, 1u);
  const dim3 gD(udiv_ceil(nRank * WARP_SZ, bD.x), 1u, 1u);
  const size_t shmD = static_cast<size_t>(0u);
  dInitV<<< gD, bD, shmD >>>(sclV);
}

void initSymbols
(double *const F,
 double *const G,
 double *const V,
 double *const S,
 double *const H,
 double *const K,
 const unsigned nRow,
 const unsigned nRank,
 const unsigned ldF,
 const unsigned ldG,
 const unsigned ldV,
 const unsigned nSwp
) throw()
{
  CUDA_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(_F), &F, sizeof(double*)));
  CUDA_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(_G), &G, sizeof(double*)));
  CUDA_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(_V), &V, sizeof(double*)));
  CUDA_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(_S), &S, sizeof(double*)));
  CUDA_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(_H), &H, sizeof(double*)));
  CUDA_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(_K), &K, sizeof(double*)));
  CUDA_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(_nRow), &nRow, sizeof(unsigned)));
  CUDA_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(_nRank), &nRank, sizeof(unsigned)));
  CUDA_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(_ldF), &ldF, sizeof(unsigned)));
  CUDA_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(_ldG), &ldG, sizeof(unsigned)));
  CUDA_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(_ldV), &ldV, sizeof(unsigned)));
  CUDA_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(_nSwp), &nSwp, sizeof(unsigned)));
  CUDA_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(_STRAT0_STEPS), &STRAT0_STEPS, sizeof(unsigned)));
  CUDA_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(_STRAT0_PAIRS), &STRAT0_PAIRS, sizeof(unsigned)));
  // copy strategy tables
  CUDA_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(_strat0), strat0, sizeof(strat0)));
  CUDA_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(_strat1), strat1, sizeof(strat1)));
}
