#include "hip/hip_runtime.h"
#include "HZ_L3.hpp"

#include "cuda_memory_helper.hpp"
#include "HZ_L.hpp"
#include "HZ_L2.hpp"

int HZ_L3
(const unsigned routine,    // IN, routine ID, <= 15, (Bb__)_2,
 // bits B, b: block-oriented (else, full-block), level 1 and 2;
 const size_t gpu,          // IN, GPU ID (0 <= gpu < gpus);
 const size_t gpus,         // IN, number of GPUs;
 const size_t mF,           // IN, number of rows of F, == 0 (mod 64);
 const size_t mG,           // IN, number of rows of G, == 0 (mod 64);
 const size_t n,            // IN, number of columns, <= min(mF, mG), == 0 (mod 32);
 const size_t n_gpu,        // IN, number of columns per GPU (2 * n_col);
 const size_t n_col,        // IN, number of columns in a block column;
 double *const hF,          // INOUT, ldhF x n_gpu host array in Fortran order;
 const size_t ldhF,         // IN, leading dimension of F, >= mF;
 double *const hG,          // INOUT, ldhG x n_gpu host array in Fortran order;
 const size_t ldhG,         // IN, leading dimension of G, >= mG;
 double *const hV,          // OUT, ldhV x n_gpu host array in Fortran order;
 const size_t ldhV,         // IN, leading dimension of V, >= n;
 double *const hS,          // OUT, the generalized singular values, optionally sorted in descending order;
 double *const hH,          // ||F_i||_2/sqrt(||F_i||_2^2 + ||G_i||_2^2);
 double *const hK,          // ||G_i||_2/sqrt(||F_i||_2^2 + ||G_i||_2^2);
 unsigned &glbSwp,          // OUT, number of sweeps at the outermost level;
 unsigned long long &glb_s, // OUT, number of rotations;
 unsigned long long &glb_b, // OUT, number of ``big'' rotations;
 double *const timing       // OUT, optional, in seconds, double[4];
) throw()
{
  switch (routine) {
  case 12:
  case 8u:
  case 4u:
  case 0u:
    break;
  default:
    return -1;
  }

  if (gpu >= gpus)
    return -2;
  if (!gpus)
    return -3;

  if (!mF)
    return -4;
  if (!mG)
    return -5;
  if (!n)
    return -6;
  if (!n_gpu)
    return -7;
  if (!n_col)
    return -8;

  if (!hF)
    return -9;
  if (ldhF < mF)
    return -10;

  if (!hG)
    return -11;
  if (ldhG < mG)
    return -12;

  if (!hV)
    return -13;
  if (ldhV < n)
    return -14;

  if (!hS)
    return -15;
  if (!hH)
    return -16;
  if (!hK)
    return -17;

  size_t lddF = mF;
  double *const dF = allocDeviceMtx<double>(lddF, mF, n_gpu, true);

  size_t lddG = mG;
  double *const dG = allocDeviceMtx<double>(lddG, mG, n_gpu, true);

  size_t lddV = n;
  double *const dV = allocDeviceMtx<double>(lddV, n, n_gpu, true);

  double *const dS = allocDeviceVec<double>(n_gpu);
  double *const dH = allocDeviceVec<double>(n_gpu);
  double *const dK = allocDeviceVec<double>(n_gpu);

  CUDA_CALL(hipDeviceSynchronize());
  CUDA_CALL(hipMemset2DAsync(dV, lddV * sizeof(double), 0, n * sizeof(double), n_gpu));
  CUDA_CALL(hipMemsetAsync(dS, 0, n_gpu * sizeof(double)));
  CUDA_CALL(hipMemsetAsync(dH, 0, n_gpu * sizeof(double)));
  CUDA_CALL(hipMemsetAsync(dK, 0, n_gpu * sizeof(double)));
  CUDA_CALL(hipDeviceSynchronize());

  unsigned alg = (routine | 1u);
  glb_s = 0ull;
  glb_b = 0ull;
  glbSwp = 0u;
  long long swp_tim = 0ll;
  stopwatch_reset(swp_tim);

  while (glbSwp < HZ_NSWEEP) {
    unsigned swp_swp = 0u;
    unsigned long long swp_rot[2u] = { 0ull, 0ull };
    for (unsigned stp = 0u; stp < STRAT2_STEPS; ++stp) {
      CUDA_CALL(hipMemcpy2DAsync(dF, lddF * sizeof(double), hF, ldhF * sizeof(double), mF * sizeof(double), n_gpu, hipMemcpyHostToDevice));
      CUDA_CALL(hipMemcpy2DAsync(dG, lddG * sizeof(double), hG, ldhG * sizeof(double), mG * sizeof(double), n_gpu, hipMemcpyHostToDevice));
      if (stp || glbSwp) {
        CUDA_CALL(hipMemcpy2DAsync(dV, lddV * sizeof(double), hV, ldhV * sizeof(double), n * sizeof(double), n_gpu, hipMemcpyHostToDevice));
      }
      CUDA_CALL(hipDeviceSynchronize());

      // TODO: determine ifc0, ifc1
      unsigned ifc0 = 0u, ifc1 = 0u, swp2 = 0u;
      unsigned long long rot2s = 0ull, rot2b = 0ull;
      const int ret = HZ_L2_gpu(alg, mF,mG,n_gpu, ifc0,ifc1, dF,lddF, dG,lddG, dV,lddV, hS,dS,dH,dK, swp2,rot2s,rot2b);
      if (ret) {
        (void)snprintf(err_msg, err_msg_size, "HZ_L2_gpu @GPU(%u) SWP(%u) STP(%u): %d", gpu, glbSwp, stp, ret);
        DIE(err_msg);
      }
      if (swp2 > swp_swp)
        swp_swp = swp2;
      swp_rot[0u] += rot2s;
      swp_rot[1u] += rot2b;

      CUDA_CALL(hipMemcpy2DAsync(hF, ldhF * sizeof(double), dF, lddF * sizeof(double), mF * sizeof(double), n_gpu, hipMemcpyDeviceToHost));
      CUDA_CALL(hipMemcpy2DAsync(hG, ldhG * sizeof(double), dG, lddG * sizeof(double), mG * sizeof(double), n_gpu, hipMemcpyDeviceToHost));
      CUDA_CALL(hipMemcpy2DAsync(hV, ldhV * sizeof(double), dV, lddV * sizeof(double), n * sizeof(double), n_gpu, hipMemcpyDeviceToHost));
      CUDA_CALL(hipDeviceSynchronize());
    }
    if (MPI_Allreduce(MPI_IN_PLACE, &swp_swp, 1, MPI_UNSIGNED, MPI_MAX, MPI_COMM_WORLD)) {
      DIE("MPI_Allreduce(swp_swp)");
    }
    if (MPI_Allreduce(MPI_IN_PLACE, swp_rot, 2, MPI_UNSIGNED_LONG_LONG, MPI_SUM, MPI_COMM_WORLD)) {
      DIE("MPI_Allreduce(swp_rot)");
    }
    glb_s += swp_rot[0u];
    glb_b += swp_rot[1u];
    ++glbSwp;

    const double tim_s = stopwatch_lap(swp_tim) * TS2S;
    if (!gpu) {
      (void)fprintf(stdout, "MAX2SWP(%2u), ROT_S(%10llu), ROT_B(%10llu), TIME(%#12.6f s)\n", swp_swp, swp_rot[0u], swp_rot[1u], tim_s);
      (void)fflush(stdout);
    }
    if (!swp_rot[1u])
      break;
  }

  CUDA_CALL(hipMemcpyAsync(hS, dS, n_gpu * sizeof(double), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpyAsync(hH, dH, n_gpu * sizeof(double), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpyAsync(hK, dK, n_gpu * sizeof(double), hipMemcpyDeviceToHost));
  CUDA_CALL(hipDeviceSynchronize());

  CUDA_CALL(hipFree(static_cast<void*>(dK)));
  CUDA_CALL(hipFree(static_cast<void*>(dH)));
  CUDA_CALL(hipFree(static_cast<void*>(dS)));
  CUDA_CALL(hipFree(static_cast<void*>(dV)));
  CUDA_CALL(hipFree(static_cast<void*>(dG)));
  CUDA_CALL(hipFree(static_cast<void*>(dF)));
  CUDA_CALL(hipDeviceSynchronize());

  return 0;
}
