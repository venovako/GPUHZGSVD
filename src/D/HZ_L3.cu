#include "hip/hip_runtime.h"
#include "HZ_L3.hpp"

#include "cuda_memory_helper.hpp"
#include "HZ_L.hpp"
#include "HZ_L2.hpp"

int HZ_L3
(const unsigned routine,    // IN, routine ID, <= 15, (Bb__)_2,
 // bits B, b: block-oriented (else, full-block), level 1 and 2;
 const size_t gpu,          // IN, GPU ID (0 <= gpu < gpus);
 const size_t gpus,         // IN, number of GPUs;
 const size_t mF,           // IN, number of rows of F, == 0 (mod 64);
 const size_t mG,           // IN, number of rows of G, == 0 (mod 64);
 const size_t n,            // IN, number of columns, <= min(mF, mG), == 0 (mod 32);
 const size_t n_gpu,        // IN, number of columns per GPU (2 * n_col);
 const size_t n_col,        // IN, number of columns in a block column;
 double *const hF,          // INOUT, ldhF x n_gpu host array in Fortran order;
 const size_t ldhF,         // IN, leading dimension of F, >= mF;
 double *const hG,          // INOUT, ldhG x n_gpu host array in Fortran order;
 const size_t ldhG,         // IN, leading dimension of G, >= mG;
 double *const hV,          // OUT, ldhV x n_gpu host array in Fortran order;
 const size_t ldhV,         // IN, leading dimension of V, >= n;
 double *const hS,          // OUT, the generalized singular values, optionally sorted in descending order;
 double *const hH,          // ||F_i||_2/sqrt(||F_i||_2^2 + ||G_i||_2^2);
 double *const hK,          // ||G_i||_2/sqrt(||F_i||_2^2 + ||G_i||_2^2);
 unsigned &glbSwp,          // OUT, number of sweeps at the outermost level;
 unsigned long long &glb_s, // OUT, number of rotations;
 unsigned long long &glb_b, // OUT, number of ``big'' rotations;
 double &timing             // OUT, in seconds;
) throw()
{
  if (routine >= 16u)
    return -1;

  if (gpu >= gpus)
    return -2;
  if (!gpus)
    return -3;

  if (!mF)
    return -4;
  if (!mG)
    return -5;
  if (!n)
    return -6;
  if (!n_gpu)
    return -7;
  if (!n_col)
    return -8;

  if (!hF)
    return -9;
  if (ldhF < mF)
    return -10;

  if (!hG)
    return -11;
  if (ldhG < mG)
    return -12;

  if (!hV)
    return -13;
  if (ldhV < n)
    return -14;

  if (!hS)
    return -15;
  if (!hH)
    return -16;
  if (!hK)
    return -17;

  size_t lddF = mF;
  double *const dF = allocDeviceMtx<double>(lddF, mF, n_gpu, true);
  if (lddF != ldhF) {
    DIE("lddF != ldhF");
  }

  size_t lddG = mG;
  double *const dG = allocDeviceMtx<double>(lddG, mG, n_gpu, true);
  if (lddG != ldhG) {
    DIE("lddG != ldhG");
  }

  size_t lddV = n;
  double *const dV = allocDeviceMtx<double>(lddV, n, n_gpu, true);
  if (lddV != ldhV) {
    DIE("lddV != ldhV");
  }

  double *const dS = allocDeviceVec<double>(n_gpu);
  double *const dH = allocDeviceVec<double>(n_gpu);
  double *const dK = allocDeviceVec<double>(n_gpu);

  CUDA_CALL(hipDeviceSynchronize());
  if (MPI_Barrier(MPI_COMM_WORLD)) {
    DIE("MPI_Barrier(init)");
  }
  long long all_tim = 0ll, swp_tim = 0ll;
  stopwatch_reset(all_tim);
  glb_s = 0ull;
  glb_b = 0ull;
  glbSwp = 0u;
  timing = 0.0;
  unsigned alg = (routine | 1u);

  CUDA_CALL(hipMemset2DAsync(dV, lddV * sizeof(double), 0, n * sizeof(double), n_gpu));
  CUDA_CALL(hipMemsetAsync(dH, 0, n_gpu * sizeof(double)));
  CUDA_CALL(hipMemsetAsync(dK, 0, n_gpu * sizeof(double)));
  CUDA_CALL(hipDeviceSynchronize());

  stopwatch_reset(swp_tim);

  while (glbSwp < HZ_NSWEEP) {
    unsigned swp_swp = 0u;
    unsigned long long swp_rot[2u] = { 0ull, 0ull };
    for (unsigned stp = 0u; stp < STRAT2_STEPS; ++stp) {
      CUDA_CALL(hipMemcpy2DAsync(dF, lddF * sizeof(double), hF, ldhF * sizeof(double), mF * sizeof(double), n_gpu, hipMemcpyHostToDevice));
      CUDA_CALL(hipMemcpy2DAsync(dG, lddG * sizeof(double), hG, ldhG * sizeof(double), mG * sizeof(double), n_gpu, hipMemcpyHostToDevice));
      if (stp || glbSwp) {
        CUDA_CALL(hipMemcpy2DAsync(dV, lddV * sizeof(double), hV, ldhV * sizeof(double), n * sizeof(double), n_gpu, hipMemcpyHostToDevice));
      }
      CUDA_CALL(hipDeviceSynchronize());

      int sp = static_cast<int>(strat2[stp][gpu][1u][0u]);
      const int tp = (sp ? ((sp < 0) ? 0 : 3) : -1);
      sp = abs(sp) - 1;

      int sq = static_cast<int>(strat2[stp][gpu][1u][1u]);
      const int tq = (sq ? ((sq < 0) ? 0 : 3) : -1);
      sq = abs(sq) - 1;

      MPI_Request r[12u] =
        { MPI_REQUEST_NULL, MPI_REQUEST_NULL, MPI_REQUEST_NULL,
          MPI_REQUEST_NULL, MPI_REQUEST_NULL, MPI_REQUEST_NULL,
          MPI_REQUEST_NULL, MPI_REQUEST_NULL, MPI_REQUEST_NULL,
          MPI_REQUEST_NULL, MPI_REQUEST_NULL, MPI_REQUEST_NULL };

      if (MPI_Irecv(hF, (ldhF * n_col), MPI_DOUBLE, MPI_ANY_SOURCE, 1, MPI_COMM_WORLD, (r + 0u))) {
        DIE("MPI_Irecv(F)p");
      }
      if (MPI_Irecv(hG, (ldhG * n_col), MPI_DOUBLE, MPI_ANY_SOURCE, 2, MPI_COMM_WORLD, (r + 1u))) {
        DIE("MPI_Irecv(G)p");
      }
      if (MPI_Irecv(hV, (ldhV * n_col), MPI_DOUBLE, MPI_ANY_SOURCE, 3, MPI_COMM_WORLD, (r + 2u))) {
        DIE("MPI_Irecv(V)p");
      }

      if (MPI_Irecv((hF + ldhF * n_col), (ldhF * n_col), MPI_DOUBLE, MPI_ANY_SOURCE, 4, MPI_COMM_WORLD, (r + 3u))) {
        DIE("MPI_Irecv(F)q");
      }
      if (MPI_Irecv((hG + ldhG * n_col), (ldhG * n_col), MPI_DOUBLE, MPI_ANY_SOURCE, 5, MPI_COMM_WORLD, (r + 4u))) {
        DIE("MPI_Irecv(G)q");
      }
      if (MPI_Irecv((hV + ldhV * n_col), (ldhV * n_col), MPI_DOUBLE, MPI_ANY_SOURCE, 6, MPI_COMM_WORLD, (r + 5u))) {
        DIE("MPI_Irecv(V)q");
      }

      const unsigned p = static_cast<unsigned>(strat2[stp][gpu][0u][0u]);
      const unsigned q = static_cast<unsigned>(strat2[stp][gpu][0u][1u]);

      const size_t ifc0 = p * n_col;
      const size_t ifc1 = q * n_col;

      unsigned swp2 = 0u;
      unsigned long long rot2s = 0ull, rot2b = 0ull;
      const int ret = HZ_L2_gpu(alg, mF,mG,n_gpu, ifc0,ifc1, dF,lddF, dG,lddG, dV,lddV, hS,dS,dH,dK, swp2,rot2s,rot2b);
      if (ret) {
        (void)snprintf(err_msg, err_msg_size, "HZ_L2_gpu @GPU(%u) SWP(%u) STP(%u): %d", gpu, glbSwp, stp, ret);
        DIE(err_msg);
      }
      if (swp2 > swp_swp)
        swp_swp = swp2;
      swp_rot[0u] += rot2s;
      swp_rot[1u] += rot2b;
   
      if (MPI_Isend(dF, (lddF * n_col), MPI_DOUBLE, sp, (1 + tp), MPI_COMM_WORLD, (r + 6u))) {
        DIE("MPI_Isend(F)p");
      }
      if (MPI_Isend(dG, (lddG * n_col), MPI_DOUBLE, sp, (2 + tp), MPI_COMM_WORLD, (r + 7u))) {
        DIE("MPI_Isend(G)p");
      }
      if (MPI_Isend(dV, (lddV * n_col), MPI_DOUBLE, sp, (3 + tp), MPI_COMM_WORLD, (r + 8u))) {
        DIE("MPI_Isend(V)p");
      }

      if (MPI_Isend((dF + lddF * n_col), (lddF * n_col), MPI_DOUBLE, sq, (1 + tq), MPI_COMM_WORLD, (r + 9u))) {
        DIE("MPI_Isend(F)q");
      }
      if (MPI_Isend((dG + lddG * n_col), (lddG * n_col), MPI_DOUBLE, sq, (2 + tq), MPI_COMM_WORLD, (r + 10u))) {
        DIE("MPI_Isend(G)q");
      }
      if (MPI_Isend((dV + lddV * n_col), (lddV * n_col), MPI_DOUBLE, sq, (3 + tq), MPI_COMM_WORLD, (r + 11u))) {
        DIE("MPI_Isend(V)q");
      }

      if (MPI_Waitall(12, r, MPI_STATUSES_IGNORE)) {
        DIE("MPI_Waitall");
      }
      CUDA_CALL(hipDeviceSynchronize());
    }
    unsigned max_swp = 0u;
    if (MPI_Allreduce(&swp_swp, &max_swp, 1, MPI_UNSIGNED, MPI_MAX, MPI_COMM_WORLD)) {
      DIE("MPI_Allreduce(max_swp)");
    }
    unsigned long long all_rot[2u] = { 0ull, 0ull };
    if (MPI_Allreduce(swp_rot, all_rot, 2, MPI_UNSIGNED_LONG_LONG, MPI_SUM, MPI_COMM_WORLD)) {
      DIE("MPI_Allreduce(all_rot)");
    }
    glb_s += all_rot[0u];
    glb_b += all_rot[1u];
    ++glbSwp;

    if (!gpu) {
      (void)fprintf(stdout, "MAX2SWP(%2u), ROT_S(%10llu), ROT_B(%10llu), TIME(%#12.6f s)\n", max_swp, all_rot[0u], all_rot[1u], (stopwatch_lap(swp_tim) * TS2S));
      (void)fflush(stdout);
    }
    if (!all_rot[1u])
      break;
  }

  // CUDA_CALL(hipMemcpy2DAsync(hF, ldhF * sizeof(double), dF, lddF * sizeof(double), mF * sizeof(double), n_gpu, hipMemcpyDeviceToHost));
  // CUDA_CALL(hipMemcpy2DAsync(hG, ldhG * sizeof(double), dG, lddG * sizeof(double), mG * sizeof(double), n_gpu, hipMemcpyDeviceToHost));
  // CUDA_CALL(hipMemcpy2DAsync(hV, ldhV * sizeof(double), dV, lddV * sizeof(double), n * sizeof(double), n_gpu, hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpyAsync(hS, dS, n_gpu * sizeof(double), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpyAsync(hH, dH, n_gpu * sizeof(double), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpyAsync(hK, dK, n_gpu * sizeof(double), hipMemcpyDeviceToHost));
  CUDA_CALL(hipDeviceSynchronize());

  if (MPI_Barrier(MPI_COMM_WORLD)) {
    DIE("MPI_Barrier(fini)");
  }
  timing = (stopwatch_lap(all_tim) * TS2S);

  CUDA_CALL(hipFree(static_cast<void*>(dK)));
  CUDA_CALL(hipFree(static_cast<void*>(dH)));
  CUDA_CALL(hipFree(static_cast<void*>(dS)));
  CUDA_CALL(hipFree(static_cast<void*>(dV)));
  CUDA_CALL(hipFree(static_cast<void*>(dG)));
  CUDA_CALL(hipFree(static_cast<void*>(dF)));
  CUDA_CALL(hipDeviceSynchronize());

  return 0;
}
