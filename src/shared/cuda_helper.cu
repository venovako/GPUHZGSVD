#include "cuda_helper.hpp"

#include "my_utils.hpp"

int configureGPUex(const int dev, const unsigned maxShMemB) throw()
{
  assert(dev >= 0);
  CUDA_CALL(hipSetDevice(dev));

  hipDeviceProp_t cdp;
  CUDA_CALL(hipGetDeviceProperties(&cdp, dev));
  const int dcc = cdp.major * 10 + cdp.minor;

  if (dcc < 30) {
    (void)snprintf(err_msg, err_msg_size, "CUDA Device %d Compute Capability %d < 30", dev, dcc);
    DIE(err_msg);
  }

  if (WARP_SZ != static_cast<unsigned>(cdp.warpSize)) {
    (void)snprintf(err_msg, err_msg_size, "CUDA Device %d has %d threads in a warp, must be %u", dev, cdp.warpSize, WARP_SZ);
    DIE(err_msg);
  }

  hipFuncCache_t cacheConfig = hipFuncCachePreferNone;
  if (maxShMemB) {
    if (maxShMemB <= 16384u) // 16 kB
      cacheConfig = hipFuncCachePreferL1;
    else if (maxShMemB <= 32768u) // 32 kB
      cacheConfig = hipFuncCachePreferEqual;
    else if (maxShMemB <= 49152u) // 48 kB
      cacheConfig = hipFuncCachePreferShared;
    else { // > 48 kB
      (void)snprintf(err_msg, err_msg_size, "Maximum shared memory requested (%u B) > 48 kB", maxShMemB);
      WARN(err_msg);
    }
  }
  CUDA_CALL(hipDeviceSetCacheConfig(cacheConfig));
  CUDA_CALL(hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte));

  return dcc;
}

int configureGPU(const int dev) throw()
{
#ifdef USE_COMPLEX
  static const unsigned maxShMemB = 49152u; // 48 kB
#else /* !USE_COMPLEX */
  static const unsigned maxShMemB = 24576u; // 24 kB
#endif /* ?USE_COMPLEX */
  return configureGPUex(dev, maxShMemB);
}

void cuda_prof_start() throw()
{
#ifdef PROFILE
  CUDA_CALL(hipProfilerStart());
#endif /* PROFILE */
}

void cuda_prof_stop() throw()
{
#ifdef PROFILE
  CUDA_CALL(hipProfilerStop());
#endif /* PROFILE */
}
