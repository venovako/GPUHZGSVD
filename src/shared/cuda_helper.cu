#include "cuda_helper.hpp"

#include "my_utils.hpp"
#ifdef PROFILE
#ifdef USE_MPI
#include "mpi_helper.hpp"
#endif // USE_MPI
#endif // PROFILE

int configureGPUex(const int dev, const unsigned maxShMemB) throw()
{
  assert(dev >= 0);
  CUDA_CALL(hipSetDevice(dev));

  hipDeviceProp_t cdp;
  CUDA_CALL(hipGetDeviceProperties(&cdp, dev));
  const int dcc = cdp.major * 10 + cdp.minor;

  if (dcc < 30) {
    (void)snprintf(err_msg, err_msg_size, "CUDA Device %d Compute Capability %d < 30", dev, dcc);
    DIE(err_msg);
  }

  if (WARP_SZ != static_cast<unsigned>(cdp.warpSize)) {
    (void)snprintf(err_msg, err_msg_size, "CUDA Device %d has %d threads in a warp, must be %u", dev, cdp.warpSize, WARP_SZ);
    DIE(err_msg);
  }

  hipFuncCache_t cacheConfig = hipFuncCachePreferNone;
  if (maxShMemB) {
    if (maxShMemB <= 16384u) // 16 kB
      cacheConfig = hipFuncCachePreferL1;
    else if (maxShMemB <= 32768u) // 32 kB
      cacheConfig = hipFuncCachePreferEqual;
    else if (maxShMemB <= 49152u) // 48 kB
      cacheConfig = hipFuncCachePreferShared;
    else { // > 48 kB
      (void)snprintf(err_msg, err_msg_size, "Maximum shared memory requested (%u B) > 48 kB", maxShMemB);
      WARN(err_msg);
    }
  }
  CUDA_CALL(hipDeviceSetCacheConfig(cacheConfig));
  CUDA_CALL(hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte));

#ifdef PROFILE
#ifndef STR1CONC
#define STR1CONC(x) #x
#else // STR1CONC
#error STR1CONC not definable externally
#endif // ?STR1CONC

#ifdef USE_MPI
#ifdef USE_COMPLEX
  (void)snprintf(err_msg, err_msg_size, "Z" STR1CONC(CVG) "_" STR1CONC(PROFILE) "_%d_%d.csv", mpi_rank, dev);
#else // !USE_COMPLEX
  (void)snprintf(err_msg, err_msg_size, "D" STR1CONC(CVG) "_" STR1CONC(PROFILE) "_%d_%d.csv", mpi_rank, dev);
#endif // ?USE_COMPLEX
#else // !USE_MPI
#ifdef USE_COMPLEX
  (void)snprintf(err_msg, err_msg_size, "Z" STR1CONC(CVG) "_" STR1CONC(PROFILE) "_%d.csv", dev);
#else // !USE_COMPLEX
  (void)snprintf(err_msg, err_msg_size, "D" STR1CONC(CVG) "_" STR1CONC(PROFILE) "_%d.csv", dev);
#endif // ?USE_COMPLEX
#endif // ?USE_MPI
  CUDA_CALL(cudaProfilerInitialize(STR1CONC(PROFILE) ".cfg", err_msg, cudaCSV));

#undef STR1CONC
#endif // PROFILE

  return dcc;
}

int configureGPU(const int dev) throw()
{
#ifdef USE_COMPLEX
  static const unsigned maxShMemB = 49152u; // 48 kB
#else // !USE_COMPLEX
  static const unsigned maxShMemB = 24576u; // 24 kB
#endif // ?USE_COMPLEX
  return configureGPUex(dev, maxShMemB);
}

void cuda_prof_start() throw()
{
#ifdef PROFILE
  CUDA_CALL(hipProfilerStart());
#endif // PROFILE
}

void cuda_prof_stop() throw()
{
#ifdef PROFILE
  CUDA_CALL(hipProfilerStop());
#endif // PROFILE
}
