#include "mpi_helper.hpp"

#include "cuda_helper.hpp"
#include "my_utils.hpp"

int mpi_size = 0;
int mpi_rank = 0;

#ifdef OMPI_MPI_H
#include <mpi-ext.h>
#endif // OMPI_MPI_H
bool mpi_cuda_aware = false;

static bool mpi_cuda() throw()
{
#if (defined(MPIX_CUDA_AWARE_SUPPORT) && MPIX_CUDA_AWARE_SUPPORT)
  return (1 == MPIX_Query_cuda_support());
#elif (defined(MVAPICH2_NUMVERSION) && (MVAPICH2_NUMVERSION >= 20000000))
  const char *const e = getenv("MV2_USE_CUDA");
  return (e && atoi(e));
#else // only OpenMPI and MVAPICH2 so far
  return false;
#endif // TODO: any other MPI?
}

int init_MPI(int *const argc, char ***const argv) throw()
{
  if (!argc)
    return -1;
  if (!argv)
    return -2;
  int f = 0, e = MPI_SUCCESS;
  if ((e = MPI_Initialized(&f)))
    return e;
  if (f)
    return MPI_SUCCESS;
  if ((e = MPI_Finalized(&f)))
    return e;
  if (f)
    return -3;
  if ((e = MPI_Init(argc, argv)))
    return e;
  if ((e = MPI_Comm_size(MPI_COMM_WORLD, &mpi_size)))
    return e;
  if ((e = MPI_Comm_rank(MPI_COMM_WORLD, &mpi_rank)))
    return e;
  mpi_cuda_aware = mpi_cuda();
  return MPI_SUCCESS;
}

int fini_MPI() throw()
{
  int f = 0, e = MPI_SUCCESS;
  if ((e = MPI_Finalized(&f)))
    return e;
  if (f)
    return MPI_SUCCESS;
  if ((e = MPI_Initialized(&f)))
    return e;
  if (!f)
    return MPI_SUCCESS;
  return MPI_Finalize();
}

#ifndef DEV_HOST_NAME_LEN
#define DEV_HOST_NAME_LEN 255u
#endif // !DEV_HOST_NAME_LEN

typedef struct {
  char host[DEV_HOST_NAME_LEN + 1u];
  int rank;
  int dev_count;
  int dev;
} dev_host;

static int dev_host_cmp(const dev_host *const a, const dev_host *const b) throw()
{
  assert(a);
  assert(b);
  if (a == b)
    return 0;
  const int hc = strcmp(a->host, b->host);
  if (hc < 0)
    return -1;
  if (hc > 0)
    return 1;
  if (a->rank < b->rank)
    return -2;
  if (a->rank > b->rank)
    return 2;
  if (a->dev_count < b->dev_count)
    return -3;
  if (a->dev_count > b->dev_count)
    return 3;
  if (a->dev < b->dev)
    return -4;
  if (a->dev > b->dev)
    return 4;
  return 0;
}

static int dev_host_get(dev_host &dh) throw()
{
  (void)memset(&dh, 0, sizeof(dh));
  dh.rank = mpi_rank;
  dh.dev = -1;
  if (gethostname(dh.host, DEV_HOST_NAME_LEN))
    return (dh.dev_count = -2);
  if (hipGetDeviceCount(&(dh.dev_count)) != hipSuccess)
    return (dh.dev_count = -1);
  return dh.dev_count;
}

static dev_host *get_dev_hosts() throw()
{
  dev_host my;
  if (dev_host_get(my) <= 0) {
    DIE("dev_host_get");
  }

  dev_host *const rcv = static_cast<dev_host*>(malloc(static_cast<unsigned>(mpi_size) * sizeof(dev_host)));
  if (!rcv) {
    DIE("out of memory");
  }

  if (MPI_Allgather(&my, static_cast<int>(sizeof(dev_host)), MPI_BYTE, rcv, static_cast<int>(sizeof(dev_host)), MPI_BYTE, MPI_COMM_WORLD)) {
    DIE("MPI_Allgather should not have failed");
  }
#ifndef NDEBUG
  if (!mpi_rank)
    (void)fprintf(stderr, "RANK,GPUS,HOSTNAME\n");
#endif // !NDEBUG
  for (int i = 0; i < mpi_size; ++i) {
#ifndef NDEBUG
    if (!mpi_rank)
      (void)fprintf(stderr, "%4d,%4d,%s\n", rcv[i].rank, rcv[i].dev_count, rcv[i].host);
#endif // !NDEBUG
    if (rcv[i].dev_count <= 0) {
      free(rcv);
      return static_cast<dev_host*>(NULL);
    }
  }

  return rcv;
}

int assign_dev2host() throw()
{
  dev_host *const dh = get_dev_hosts();
  if (!dh)
    return -1;

  if (mpi_size > 1)
    qsort(dh, static_cast<size_t>(mpi_size), sizeof(dev_host), (int (*)(const void*, const void*))dev_host_cmp);

  int dev = -3;
  if (!mpi_rank)
    (void)fprintf(stderr, "RANK,GPUS,LGPU,HOSTNAME\n");
  dh[0].dev = 0;
  if (!mpi_rank)
    (void)fprintf(stderr, "%4d,%4d,%4d,%s\n", dh[0].rank, dh[0].dev_count, dh[0].dev, dh[0].host);
  if (dh[0].rank == mpi_rank)
    dev = dh[0].dev;

  for (int i = 1; i < mpi_size; ++i) {
    int err = 0;
    if (!strcmp(dh[i].host, dh[i-1].host)) {
      // inconsistent data
      if (dh[i].dev_count != dh[i-1].dev_count)
        err = -2;
      dh[i].dev = (dh[i-1].dev + 1);
      // more processes than devices per host, wrap around
      if (dh[i].dev >= dh[i].dev_count)
        dh[i].dev = 0;
    }
    else
      dh[i].dev = 0;
    if (!mpi_rank)
      (void)fprintf(stderr, "%4d,%4d,%4d,%s\n", dh[i].rank, dh[i].dev_count, dh[i].dev, dh[i].host);
    if (err) {
      dev = err;
      goto end;
    }
    if (dh[i].rank == mpi_rank)
      dev = dh[i].dev;
  }

 end:
  (void)fflush(stderr);
  free(dh);
  return dev;
}
