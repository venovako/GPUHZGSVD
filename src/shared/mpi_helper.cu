#include "mpi_helper.hpp"

#include "cuda_helper.hpp"
#include "my_utils.hpp"

#ifdef OMPI_MPI_H
#include <mpi-ext.h>
#endif // OMPI_MPI_H

int mpi_size = 0;
int mpi_rank = 0;
bool mpi_cuda_aware = false;

#ifdef USE_MPI_IO
#ifdef USE_COMPLEX
MPI_Datatype DT_V112D = MPI_DATATYPE_NULL;
#endif // USE_COMPLEX
#endif // USE_MPI_IO

static bool mpi_cuda() throw()
{
#if (defined(MPIX_CUDA_AWARE_SUPPORT) && MPIX_CUDA_AWARE_SUPPORT)
  return (1 == MPIX_Query_cuda_support());
#elif (defined(MVAPICH2_NUMVERSION) && (MVAPICH2_NUMVERSION >= 20000000))
  const char *const e = getenv("MV2_USE_CUDA");
  return (e && atoi(e));
#else // only OpenMPI and MVAPICH2 so far
  return false;
#endif // TODO: any other MPI?
}

int init_MPI(int *const argc, char ***const argv) throw()
{
  if (!argc)
    return -1;
  if (!argv)
    return -2;
  int i = 0, f = 0, e = MPI_SUCCESS;
  if ((e = MPI_Initialized(&i)))
    return e;
  if (i)
    return MPI_SUCCESS;
  if ((e = MPI_Finalized(&f)))
    return e;
  if (f)
    return -3;
  if ((e = MPI_Init(argc, argv)))
    return e;
  if ((e = MPI_Comm_size(MPI_COMM_WORLD, &mpi_size)))
    return e;
  if ((e = MPI_Comm_rank(MPI_COMM_WORLD, &mpi_rank)))
    return e;
  mpi_cuda_aware = mpi_cuda();
#ifdef USE_MPI_IO
#ifdef USE_COMPLEX
  if ((e = MPI_Type_vector(1, 1, 2, MPI_DOUBLE, &DT_V112D)))
    return e;
  if ((e = MPI_Type_commit(&DT_V112D)))
    return e;
#endif // USE_COMPLEX
#endif // USE_MPI_IO
  return MPI_SUCCESS;
}

int fini_MPI() throw()
{
  int f = 0, e = MPI_SUCCESS;
  if ((e = MPI_Finalized(&f)))
    return e;
  if (f)
    return MPI_SUCCESS;
#ifdef USE_MPI_IO
#ifdef USE_COMPLEX
  if ((e = MPI_Type_free(&DT_V112D)))
    return e;
#endif // USE_COMPLEX
#endif // USE_MPI_IO
  return MPI_Finalize();
}

#ifndef DEV_HOST_NAME_LEN
#define DEV_HOST_NAME_LEN 255u
#endif // !DEV_HOST_NAME_LEN

typedef struct {
  char host[DEV_HOST_NAME_LEN + 1u];
  int rank;
  int dev_count;
  int dev;
} dev_host;

static int dev_host_cmp(const dev_host *const a, const dev_host *const b) throw()
{
  assert(a);
  assert(b);
  if (a == b)
    return 0;
  const int hc = strcmp(a->host, b->host);
  if (hc < 0)
    return -1;
  if (hc > 0)
    return 1;
  if (a->rank < b->rank)
    return -2;
  if (a->rank > b->rank)
    return 2;
  if (a->dev_count < b->dev_count)
    return -3;
  if (a->dev_count > b->dev_count)
    return 3;
  if (a->dev < b->dev)
    return -4;
  if (a->dev > b->dev)
    return 4;
  return 0;
}

static int dev_host_get(dev_host &dh) throw()
{
  (void)memset(&dh, 0, sizeof(dh));
  dh.rank = mpi_rank;
  dh.dev = -1;
  if (gethostname(dh.host, DEV_HOST_NAME_LEN))
    return (dh.dev_count = -2);
  if (hipGetDeviceCount(&(dh.dev_count)) != hipSuccess)
    return (dh.dev_count = -1);
  return dh.dev_count;
}

static dev_host *get_dev_hosts() throw()
{
  dev_host my;
  if (dev_host_get(my) <= 0)
    (void)fprintf(stderr, "Cannot query the host information (%d)\n", my.dev_count);

  dev_host *const rcv = static_cast<dev_host*>(malloc(static_cast<unsigned>(mpi_size) * sizeof(dev_host)));
  SYSP_CALL(rcv);

  SYSI_CALL(MPI_Allgather(&my, static_cast<int>(sizeof(dev_host)), MPI_BYTE, rcv, static_cast<int>(sizeof(dev_host)), MPI_BYTE, MPI_COMM_WORLD));

  if (!mpi_rank)
    (void)fprintf(stderr, "RANK,GPUS,HOSTNAME\n");
  for (int i = 0; i < mpi_size; ++i) {
    if (!mpi_rank)
      (void)fprintf(stderr, "%4d,%4d,%s\n", rcv[i].rank, rcv[i].dev_count, rcv[i].host);
    if (rcv[i].dev_count <= 0) {
      free(rcv);
      return static_cast<dev_host*>(NULL);
    }
  }

  return rcv;
}

int assign_dev2host() throw()
{
  dev_host *const dh = get_dev_hosts();
  if (!dh)
    return -1;

  if (mpi_size > 1)
    qsort(dh, static_cast<size_t>(mpi_size), sizeof(dev_host), (int (*)(const void*, const void*))dev_host_cmp);

  int dev = -3;
  if (!mpi_rank)
    (void)fprintf(stderr, "\nRANK,GPUS,LGPU,HOSTNAME\n");
  dh[0].dev = 0;
  if (!mpi_rank)
    (void)fprintf(stderr, "%4d,%4d,%4d,%s\n", dh[0].rank, dh[0].dev_count, dh[0].dev, dh[0].host);
  if (dh[0].rank == mpi_rank)
    dev = dh[0].dev;

  for (int i = 1; i < mpi_size; ++i) {
    int err = 0;
    if (!strcmp(dh[i].host, dh[i-1].host)) {
      // inconsistent data
      if (dh[i].dev_count != dh[i-1].dev_count)
        err = -2;
      dh[i].dev = (dh[i-1].dev + 1);
      // more processes than devices per host, wrap around
      if (dh[i].dev >= dh[i].dev_count)
        dh[i].dev = 0;
    }
    else
      dh[i].dev = 0;
    if (!mpi_rank)
      (void)fprintf(stderr, "%4d,%4d,%4d,%s\n", dh[i].rank, dh[i].dev_count, dh[i].dev, dh[i].host);
    if (err) {
      dev = err;
      goto end;
    }
    if (dh[i].rank == mpi_rank)
      dev = dh[i].dev;
  }

 end:
  free(dh);
  return dev;
}
